#include <assert.h>

#include <algorithm>
#include <chrono>
#include <iostream>
#include <istream>
#include <vector>

#include "gpuknn/knncuda_tools.cuh"
#include "gpuknn/knnmerge.cuh"
#include "gpuknn/nndescent.cuh"
#include "tools/distfunc.hpp"
#include "tools/filetool.hpp"
#include "xmuknn.h"

using namespace std;
using namespace xmuknn;

void Evaluate(const string &data_path, const string &ground_truth_path) {
  string cmd = "python3 -u \"/home/hwang/codes/GPU_KNNG/tools/evaluate.py\"";
  cmd += " ";
  cmd += data_path;
  cmd += " ";
  cmd += ground_truth_path;
  int re = system(cmd.c_str());
}

void TestCUDANNDescent() {
  int k = 30;
  // string out_path = FileTool::GetOutPath();

  // string base_path
  //     = "/home/hwang//data/sift10k/sift10k.txt";
  // string out_path
  //     = "/home/hwang/data/result/sift10k_knng_k64.txt";
  // string ground_truth_path
  //     = "/home/hwang//data/sift10k/sift10k_groundtruth_self.txt";

  // string base_path = "/home/hwang//data/sift100k/sift100k.txt";
  // string out_path = "/home/hwang/data/result/sift100k_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang//data/sift100k/sift100k_groundtruth_self.txt";

  // string base_path = "/home/hwang/data/glove1m/glove1m_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove1m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove1m/glove1m_gold_knn40.txt";

  // string base_path = "/home/hwang//data/sift1m/sift1m.txt";
  // string out_path = "/home/hwang/data/result/sift1m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang//data/sift1m/sift1m_gold_knn40_sorted.txt";

  // string base_path = "/home/hwang//data/sift10m/sift10m.txt";
  // string out_path = "/home/hwang/data/result/sift10m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift10m/sift10m_gold_knn40.txt";

  // string base_path = "/home/hwang/data/glove100k/glove100k_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove100k_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove100k/glove100k_self_ground_truth.txt";

  string base_path = "/home/hwang/data/sift1m/sift_base.fvecs";
  string out_path = "/home/hwang/data/result/sift1m_knng_k64.txt";
  string ground_truth_path =
      "/home/hwang//data/sift1m/sift1m_gold_knn40_sorted.txt";

  // string base_path = "/home/hwang/data/sift5m/sift5m.fvecs";
  // string out_path = "/home/hwang/data/result/sift5m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift1m/sift1m_gold_knn40_sorted.txt";

  auto out = ofstream(out_path);
  if (!out.is_open()) {
    cerr << "Output file is not opened!" << endl;
    return;
  }

  float *vectors;
  int vecs_size, vecs_dim;
  // FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);
  FileTool::ReadFVecs(base_path, &vectors, &vecs_size, &vecs_dim);

  auto knn_graph = gpuknn::NNDescent(vectors, vecs_size, vecs_dim, 6);

  // out << knn_graph.size() << " " << k << endl;
  // for (int i = 0; i < knn_graph.size(); i++) {
  //   const auto &x = knn_graph[i];
  //   out << i << " " << x.size() << " ";
  //   for (auto y : x) {
  //     // out << y.distance() << " " << y.label() << "\t";
  //     assert(y.label() != i);
  //     out << y.label() << "\t";
  //   }
  //   out << endl;
  // }
  // out.close();
  // Evaluate(out_path, ground_truth_path);
  delete[] vectors;
  return;
}

void DivideData(float *vectors, const int dim, float **vectors_first_ptr,
                const int vectors_first_size, float **vectors_second_ptr,
                const int vectors_second_size) {
  float *&vectors_first = *vectors_first_ptr;
  float *&vectors_second = *vectors_second_ptr;
  vectors_first = new float[vectors_first_size * dim];
  vectors_second = new float[vectors_second_size * dim];
  for (int i = 0; i < vectors_first_size; i++) {
    for (int j = 0; j < dim; j++) {
      vectors_first[i * dim + j] = vectors[i * dim + j];
    }
  }
  for (int i = 0; i < vectors_second_size; i++) {
    for (int j = 0; j < dim; j++) {
      vectors_second[i * dim + j] = vectors[(i + vectors_first_size) * dim + j];
    }
  }
  return;
}

void TestCUDAMerge() {
  string base_path = "/home/hwang/data/sift100k/sift100k.txt";
  string out_path = "/home/hwang/data/result/sift100k_knng_k64_merged.txt";
  string ground_truth_path =
      "/home/hwang//data/sift100k/sift100k_groundtruth_self.txt";
  // string base_path = "/home/hwang/data/glove100k/glove100k_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove100k_knng_k64_merged.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove100k/glove100k_self_ground_truth.txt";
  // string base_path = "/home/hwang/data/sift1m/sift1m.txt";
  // string out_path = "/home/hwang/data/result/sift1m_knng_k64_merged.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift1m/sift1m_gold_knn40_sorted.txt";
  float *vectors;
  int vecs_size, vecs_dim;
  FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);
  int vectors_first_size = vecs_size / 2;
  int vectors_second_size = vecs_size - vectors_first_size;
  float *vectors_first, *vectors_second;
  DivideData(vectors, vecs_dim, &vectors_first, vectors_first_size,
             &vectors_second, vectors_second_size);
  float *vectors_first_dev, *vectors_second_dev;
  hipMalloc(&vectors_first_dev,
             (size_t)vectors_first_size * vecs_dim * sizeof(float));
  hipMalloc(&vectors_second_dev,
             (size_t)vectors_second_size * vecs_dim * sizeof(float));
  hipMemcpy(vectors_first_dev, vectors_first,
             (size_t)vectors_first_size * vecs_dim * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(vectors_second_dev, vectors_second,
             (size_t)vectors_second_size * vecs_dim * sizeof(float),
             hipMemcpyHostToDevice);

  NNDElement *knngraph_first_dev, *knngraph_second_dev;
  gpuknn::NNDescent(&knngraph_first_dev, vectors_first_dev, vectors_first_size,
                    vecs_dim);
  gpuknn::NNDescent(&knngraph_second_dev, vectors_second_dev,
                    vectors_second_size, vecs_dim);

  NNDElement *knngraph_merged_dev;
  float *vectors_merged_dev;
  gpuknn::KNNMerge(&knngraph_merged_dev, &vectors_merged_dev, vectors_first_dev,
                   vectors_first_size, knngraph_first_dev, vectors_second_dev,
                   vectors_second_size, knngraph_second_dev);

  vector<vector<NNDElement>> knngraph_host;
  // ToHostKNNGraph(&knngraph_host, knngraph_first_dev, vectors_first_size,
  //                NEIGHB_NUM_PER_LIST);
  // OutputHostKNNGraph(knngraph_host,
  //                    "/home/hwang/codes/GPU_KNNG/results/graph_a.txt");
  // ToHostKNNGraph(&knngraph_host, knngraph_second_dev, vectors_second_size,
  //                NEIGHB_NUM_PER_LIST);
  // OutputHostKNNGraph(knngraph_host,
  //                    "/home/hwang/codes/GPU_KNNG/results/graph_b.txt");
  ToHostKNNGraph(&knngraph_host, knngraph_merged_dev,
                 vectors_first_size + vectors_second_size, NEIGHB_NUM_PER_LIST);
  OutputHostKNNGraph(knngraph_host, out_path);
  Evaluate(out_path, ground_truth_path);
  hipFree(vectors_merged_dev);
  hipFree(knngraph_merged_dev);
  hipFree(knngraph_first_dev);
  hipFree(knngraph_second_dev);
  delete[] vectors;
  delete[] vectors_first;
  delete[] vectors_second;
}

void TestFileTools() {
  // float *vectors;
  // int num;
  // int dim;
  // auto start = chrono::steady_clock::now();
  // FileTool::ReadFVecs("/home/hwang/data/sift1m/sift_base.fvecs", &vectors, &num,
  //                     &dim);
  // auto end = chrono::steady_clock::now();
  // float time_cost =
  //     (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
  //         .count() /
  //     1e6;
  // cerr << time_cost << endl;
  // float *gt_vectors;
  // int gt_num;
  // int gt_dim;
  // FileTool::ReadVecs(gt_vectors, gt_num, gt_dim,
  //                    "/home/hwang/data/sift1m/sift1m.txt");
  // for (int i = 0; i < num; i++) {
  //   for (int j = 0; j < dim; j++) {
  //     int pos = i * dim + j;
  //     assert(vectors[pos] == gt_vectors[pos]);
  //   }
  // }
  // delete[] vectors;
  // delete[] gt_vectors;

  // float *vectors;
  // int num;
  // int dim;
  // auto start = chrono::steady_clock::now();
  // FileTool::ReadFVecs("/media/hwang_data/deep1b/base_00", &vectors, &num,
  //                     &dim);
  // auto end = chrono::steady_clock::now();
  // float time_cost =
  //     (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
  //         .count() /
  //     1e6;
  // FileTool::WriteFVecs("./test.fvecs", vectors, num, dim);
  // delete[] vectors;

  // int *vectors;
  // int num;
  // int dim;
  // FileTool::ReadIVecs("/home/hwang/data/sift1m/sift_groundtruth.ivecs",
  //                     &vectors, &num, &dim);
  // cerr << num << " " << dim << endl;
  // for (int i = 0; i < 200; i++) {
  //   cerr << vectors[i] << " ";
  // } cerr << endl;
  
  // float *vectors;
  // int num;
  // int dim;
  // auto start = chrono::steady_clock::now();
  // FileTool::ReadFVecs("/home/hwang/data/sift10m/sift10m.fvecs", &vectors, &num,
  //                     &dim, 5000000);
  // auto end = chrono::steady_clock::now();
  // float time_cost =
  //     (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
  //         .count() /
  //     1e6;
  // FileTool::WriteFVecs("/home/hwang/data/sift5m/sift5m.fvecs", vectors, num, dim);
  // delete[] vectors;

  float *vectors;
  int start_pos = 900000;
  int num = 100000;
  int dim;
  auto start = chrono::steady_clock::now();
  FileTool::ReadFVecs("/home/hwang/data/sift1m/sift_base.fvecs", &vectors,
                      &dim, start_pos, num);
  auto end = chrono::steady_clock::now();
  float time_cost =
      (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() /
      1e6;
  float *gt_vectors;
  int gt_num, gt_dim;
  FileTool::ReadVecs(gt_vectors, gt_num, gt_dim,
                     "/home/hwang/data/sift1m/sift1m.txt");
  cerr << num << " " << dim << endl;
  for (int i = start_pos; i < start_pos + num; i++) {
    for (int j = 0; j < dim; j++) {
      assert(vectors[(i-start_pos) * dim + j] == gt_vectors[i * dim + j]);
    }
  }
  delete[] vectors;
}

void TestMemoryManager() {
  PredPeakGPUMemory(5000000, 128, 64, 32);
}

int main() {
  // UnitTest();
  // TestKNNAlgorithm();
  TestCUDANNDescent();
  // TestFileTools();
  // TestMemoryManager();
  // TestCUDAMerge();
  // TestTiledDistanceCompare();
  // TestCUDADistance();
  // TestCUDASearch();
  // TestCUDANewSearch();
  // TestCUDAPriorityQueue();
  // GetRGraph();
}