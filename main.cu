#include <assert.h>

#include <algorithm>
#include <chrono>
#include <iostream>
#include <istream>
#include <vector>

#include "gpuknn/gen_large_knngraph.cuh"
#include "gpuknn/knncuda_tools.cuh"
#include "gpuknn/knnmerge.cuh"
#include "gpuknn/nndescent.cuh"
#include "tools/distfunc.hpp"
#include "tools/filetool.hpp"
#include "tools/knndata_manager.hpp"
#include "xmuknn.h"

using namespace std;
using namespace xmuknn;

void Evaluate(const string &data_path, const string &ground_truth_path) {
  string cmd = "python3 -u \"/home/hwang/codes/GPU_KNNG/tools/evaluate.py\"";
  cmd += " ";
  cmd += data_path;
  cmd += " ";
  cmd += ground_truth_path;
  int re = system(cmd.c_str());
}

void TestCUDANNDescent() {
  int k = 30;
  // string out_path = FileTool::GetOutPath();

  // string base_path
  //     = "/home/hwang//data/sift10k/sift10k.txt";
  // string out_path
  //     = "/home/hwang/data/result/sift10k_knng_k64.txt";
  // string ground_truth_path
  //     = "/home/hwang//data/sift10k/sift10k_groundtruth_self.txt";

  // string base_path = "/home/hwang//data/sift100k/sift100k.txt";
  // string out_path = "/home/hwang/data/result/sift100k_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang//data/sift100k/sift100k_groundtruth_self.txt";

  // string base_path = "/home/hwang/data/glove1m/glove1m_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove1m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove1m/glove1m_gold_knn40.txt";

  // string base_path = "/home/hwang//data/sift1m/sift1m.txt";
  // string out_path = "/home/hwang/data/result/sift1m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang//data/sift1m/sift1m_gold_knn40_sorted.txt";

  // string base_path = "/home/hwang//data/sift10m/sift10m.fvecs";
  // string out_path = "/home/hwang/data/result/sift10m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift10m/sift10m_gold_knn40.txt";

  // string base_path = "/home/hwang/data/glove100k/glove100k_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove100k_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove100k/glove100k_self_ground_truth.txt";

  string base_path = "/home/hwang/data/sift1m/sift_base.fvecs";
  string out_path = "/home/hwang/data/result/sift1m_knng_k64.txt";
  string ground_truth_path =
      "/home/hwang//data/sift1m/sift1m_gold_knn40_sorted.txt";

  // string base_path = "/home/hwang/data/sift5m/sift5m.fvecs";
  // string out_path = "/home/hwang/data/result/sift5m_knng_k64.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift1m/sift1m_gold_knn40_sorted.txt";

  auto out = ofstream(out_path);
  if (!out.is_open()) {
    cerr << "Output file is not opened!" << endl;
    return;
  }

  float *vectors;
  int vecs_size, vecs_dim;
  // FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);
  FileTool::ReadBinaryVecs(base_path, &vectors, &vecs_size, &vecs_dim);
  auto knn_graph = gpuknn::NNDescent(vectors, vecs_size, vecs_dim, 6);

  // vector<float> vectors_vec(16000000ul * 128ul);
  // float *vectors;
  // int vecs_size, vecs_dim;
  // // FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);
  // FileTool::ReadBinaryVecs(base_path, &vectors, &vecs_size, &vecs_dim);
  // for (size_t i = 0; i < 16000000ul * 128ul; i++) {
  //   vectors_vec[i] = vectors[i % 10000000];
  // }
  // vecs_size = 16000000;
  // auto knn_graph =
  //     gpuknn::NNDescent(vectors_vec.data(), vecs_size, vecs_dim, 6);

  out << knn_graph.size() << " " << k << endl;
  for (int i = 0; i < knn_graph.size(); i++) {
    const auto &x = knn_graph[i];
    out << i << " " << x.size() << " ";
    for (auto y : x) {
      // out << y.distance() << ", " << y.label() << "\t";
      // assert(y.label() != i);
      out << y.label() << "\t";
    }
    out << endl;
  }
  out.close();
  Evaluate(out_path, ground_truth_path);
  delete[] vectors;
  return;
}

void DivideData(float *vectors, const int dim, float **vectors_first_ptr,
                const int vectors_first_size, float **vectors_second_ptr,
                const int vectors_second_size) {
  float *&vectors_first = *vectors_first_ptr;
  float *&vectors_second = *vectors_second_ptr;
  vectors_first = new float[vectors_first_size * dim];
  vectors_second = new float[vectors_second_size * dim];
  for (int i = 0; i < vectors_first_size; i++) {
    for (int j = 0; j < dim; j++) {
      vectors_first[i * dim + j] = vectors[i * dim + j];
    }
  }
  for (int i = 0; i < vectors_second_size; i++) {
    for (int j = 0; j < dim; j++) {
      vectors_second[i * dim + j] = vectors[(i + vectors_first_size) * dim + j];
    }
  }
  return;
}

void TestCUDAMerge() {
  // string base_path = "/home/hwang/data/sift100k/sift100k.txt";
  // string out_path = "/home/hwang/data/result/sift100k_knng_k64_merged.txt";
  // string ground_truth_path =
  //     "/home/hwang//data/sift100k/sift100k_groundtruth_self.txt";
  // string base_path = "/home/hwang/data/glove100k/glove100k_norm_base.txt";
  // string out_path = "/home/hwang/data/result/glove100k_knng_k64_merged.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/glove100k/glove100k_self_ground_truth.txt";
  // string base_path = "/home/hwang/data/sift1m/sift1m.fvecs";
  // string out_path = "/home/hwang/data/result/sift1m_knng_k64_merged.txt";
  // string ground_truth_path =
  //     "/home/hwang/data/sift1m/sift1m_gold_knn40_sorted.txt";
  string base_path = "/home/hwang//data/sift10m/sift10m.fvecs";
  string out_path = "/home/hwang/data/result/sift10m_knng_k64.txt";
  string ground_truth_path =
      "/home/hwang/data/sift10m/sift10m_gold_knn40.txt";
  float *vectors;
  int vecs_size, vecs_dim;
  FileTool::ReadBinaryVecs(base_path, &vectors, &vecs_size, &vecs_dim);
  // FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);
  int vectors_first_size = vecs_size / 2;
  int vectors_second_size = vecs_size - vectors_first_size;
  float *vectors_first, *vectors_second;
  DivideData(vectors, vecs_dim, &vectors_first, vectors_first_size,
             &vectors_second, vectors_second_size);
  float *vectors_first_dev, *vectors_second_dev;
  hipMalloc(&vectors_first_dev,
             (size_t)vectors_first_size * vecs_dim * sizeof(float));
  hipMalloc(&vectors_second_dev,
             (size_t)vectors_second_size * vecs_dim * sizeof(float));
  hipMemcpy(vectors_first_dev, vectors_first,
             (size_t)vectors_first_size * vecs_dim * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(vectors_second_dev, vectors_second,
             (size_t)vectors_second_size * vecs_dim * sizeof(float),
             hipMemcpyHostToDevice);

  NNDElement *knngraph_first_dev, *knngraph_second_dev;
  gpuknn::NNDescent(&knngraph_first_dev, vectors_first_dev, vectors_first_size,
                    vecs_dim);
  gpuknn::NNDescent(&knngraph_second_dev, vectors_second_dev,
                    vectors_second_size, vecs_dim);
  NNDElement *knngraph_merged_dev;
  gpuknn::KNNMerge(&knngraph_merged_dev, vectors_first_dev, vectors_first_size,
                   knngraph_first_dev, vectors_second_dev, vectors_second_size,
                   knngraph_second_dev, true);

  vector<vector<NNDElement>> knngraph_host;
  // ToHostKNNGraph(&knngraph_host, knngraph_first_dev, vectors_first_size,
  //                NEIGHB_NUM_PER_LIST);
  // OutputHostKNNGraph(knngraph_host,
  //                    "/home/hwang/codes/GPU_KNNG/results/graph_a.txt");
  // ToHostKNNGraph(&knngraph_host, knngraph_second_dev, vectors_second_size,
  //                NEIGHB_NUM_PER_LIST);
  // OutputHostKNNGraph(knngraph_host,
  //                    "/home/hwang/codes/GPU_KNNG/results/graph_b.txt");
  ToHostKNNGraph(&knngraph_host, knngraph_merged_dev,
                 vectors_first_size + vectors_second_size, NEIGHB_NUM_PER_LIST);
  OutputHostKNNGraph(knngraph_host, out_path);
  Evaluate(out_path, ground_truth_path);
  hipFree(knngraph_merged_dev);
  hipFree(knngraph_first_dev);
  hipFree(knngraph_second_dev);
  delete[] vectors;
  delete[] vectors_first;
  delete[] vectors_second;
}

void TestFileTools() {
  // float *vectors;
  // int num, dim;
  // FileTool::ReadBinaryVecs("/home/hwang/data/sift10m/sift10m.fvecs", &vectors, &num,
  //                          &dim);
  // cerr << num << endl;
  // for (int i = 0; i < 512; i++) {
  //   if (i % 128 == 0) puts("\n");
  //   printf("%.0f ", vectors[i]);
  // } puts("");
  // delete[] vectors;

  // float *vectors;
  // int num = 50000000, dim;
  // FileTool::ReadBinaryVecs("/home/hwang/data/deep1b/deep1b.fvecs", &vectors,
  //                          &dim, 50000000, num);
  // for (int i = 0; i < 128; i++) {
  //   if (i % 128 == 0) puts("\n");
  //   printf("%f ", vectors[i]);
  // } puts("");
  // FileTool::WriteBinaryVecs("/home/hwang/data/deep100m/deep100m_2.fvecs", vectors,
  //                           num, dim);
  // delete[] vectors;

  float *vectors;
  int num, dim;
  FileTool::ReadVecs(vectors, num, dim,
                     "/home/hwang/data/new_yfcc1m/yfcc1m_txt.txt");
  FileTool::WriteBinaryVecs("/home/hwang/data/new_yfcc1m/yfcc1m.fvecs", vectors,
                            num, dim);
}

void TestMemoryManager() {
  PredPeakGPUMemory(16000000, 128, 32, 32, false);
}

void TestDataManager() {
  KNNDataManager data_manager("/home/hwang/data/deep1b/deep1b");
  data_manager.CheckStatus();
  string cmd;
  while (cin >> cmd) {
    auto start = chrono::steady_clock::now();
    if (cmd == "add") {
      int id;
      cin >> id;
      data_manager.ActivateShard(id);
    } else if (cmd == "del"){
      int id;
      cin >> id;
      data_manager.DiscardShard(id);
    } else if (cmd == "qry") {
      int id;
      cin >> id;
      for (int i = 0; i < 96; i++) {
        cerr << data_manager.GetVectors(id)[i] << " ";
      } cerr << endl;
    } 
    else {
      cout << "Unknown command" << endl;
    }
    auto end = chrono::steady_clock::now();
    float time_cost =
        (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count() /
        1e6;
    cout << "Time costs: " << time_cost << endl;
    data_manager.OutPutActiveIds();
  }
}

void TestConstructLargeKNNGraph() {
  GenLargeKNNGraph("/home/hwang/data/sift1m/sift1m",
                   "/home/hwang/data/result/sift1m.kgraph", 64);
}

void CheckKNNGraph() {
  NNDElement *knn_graph;
  int num, k;
  FileTool::ReadBinaryVecs("/home/hwang/data/sift10m/sift10m.kgraph", &knn_graph,
                           &num, &k);
  cout << num << " " << k << endl;
  int id;
  while (cin >> id) {
    for (int i = 0; i < k; i++) {
      printf("(%f, %d) ", knn_graph[id * k + i].label(),
             knn_graph[id * k + i].distance());
    } puts("");
  }
  delete[] knn_graph;
}

void TxtToIVecs() {
  string in_path = "/home/hwang/data/sift1m/sift1m_gold_knn40_sorted.txt";
  string out_path = "/home/hwang/data/sift1m/sift1m_knngraph_k40.ivecs";
  ifstream in(in_path);
  int graph_size, dim;
  in >> graph_size >> dim;
  int *knn_graph = new int[graph_size * dim];
  for (int i = 0; i < graph_size; i++) {
    int id, neighb_num;
    in >> id >> neighb_num;
    for (int j = 0; j < neighb_num; j++) {
      int nb_id;
      in >> nb_id;
      knn_graph[i * neighb_num + j] = nb_id;
    }
  }
  FileTool::WriteBinaryVecs(out_path, knn_graph, graph_size, dim);
  delete[] knn_graph;
}

int main() {
  // UnitTest();
  // TestKNNAlgorithm();
  // TestCUDANNDescent();
  // TestDataManager();
  TestConstructLargeKNNGraph();
  // TxtToIVecs();
  // TestFileTools();
  // TestMemoryManager();
  // TestCUDAMerge();
  // TestTiledDistanceCompare();
  // TestCUDADistance();
  // TestCUDASearch();
  // TestCUDANewSearch();
  // TestCUDAPriorityQueue();
  // GetRGraph();
  // CheckKNNGraph();
}