#include <vector>
#include <iostream>
#include <istream>
#include <algorithm>
#include <chrono>

#include "xmuknn.h"
#include "tools/filetool.hpp"
#include "tools/distfunc.hpp"

#include "gpuknn/nndescent.cuh"
#include "gpuknn/unittest.cu"

using namespace std;
using namespace xmuknn;

void evaluate(const string &data_path, const string &ground_truth_path) {
    string cmd = "python3 -u \"/home/hwang/codes/GPU_KNNG/tools/evaluate.py\"";
    cmd += " "; cmd += data_path;
    cmd += " "; cmd += ground_truth_path;
    system(cmd.c_str());
}

struct KNNItem {
    int id;
    bool visited = false;
    KNNItem(int id, bool visited) :id(id), visited(visited) {}
};

void TestCUDANNDescent() {
    int k = 30;
    // string out_path = FileTool::GetOutPath();

    // string base_path 
    //     = "/mnt/d/cache/hwang_data/data/sift10k/sift10k.txt";
    // string out_path 
    //     = "/mnt/d/cache/hwang_data/data/result/sift10k_knng_k32.txt";
    // string ground_truth_path 
    //     = "/mnt/d/cache/hwang_data/data/sift10k/sift10k_groundtruth_self.txt";

    string base_path 
        = "/mnt/d/cache/hwang_data/data/sift100k/sift100k.txt";
    string out_path 
        = "/mnt/d/cache/hwang_data/data/result/sift100k_knng_k30.txt";
    string ground_truth_path 
        = "/mnt/d/cache/hwang_data/data/sift100k/sift100k_groundtruth_self.txt";

    // string base_path 
    //     = "/mnt/d/cache/hwang_data/data/sift1m/sift1m.txt";
    // string out_path 
    //     = "/mnt/d/cache/hwang_data/data/result/sift1m_knng_k30.txt";
    // string ground_truth_path 
    //     = "/mnt/d/cache/hwang_data/data/sift1m/sift1m_gold_knn40_sorted.txt";

    auto out = ofstream(out_path);
    if (!out.is_open()) {
        cerr << "Output file is not opened!" << endl;
        return;
    }

    float* vectors;
    int vecs_size, vecs_dim;
    FileTool::ReadVecs(vectors, vecs_size, vecs_dim, base_path);

    auto start = chrono::steady_clock::now();
    auto knn_graph = gpuknn::NNDescent(vectors, vecs_size, vecs_dim);
    auto end = chrono::steady_clock::now();

    out << knn_graph.size() << " " << k << endl;
    for (int i = 0; i < knn_graph.size(); i++) {
        const auto &x = knn_graph[i];
        out << i << " " << x.size() << " ";
        for (auto y : x) {
            // out << y.distance << " " << y.id << "\t";
            out << y.id << "\t";
        } out << endl;
    }
    out.close();
    cerr << "GPU NNDescent costs: "
         << (float)chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6
         << endl;
    evaluate(out_path, ground_truth_path);
}

void TestCUDAASM() {
    int x, pos;
    while (cin >> x) {
        cerr << "start" << endl;
        for (int i = 0; i < 32; i++) {
            TestASMKernel<<<dim3(1), dim3(1)>>> (x, i);
            hipDeviceSynchronize();
        } cerr << endl;
        cerr << "end" << endl;
    }
}

void UnitTest() {
    // TestKNNListInsert();
    // TestCUDAASM();
    TestLocalListUpdate();
}

int main() {
    UnitTest();
    //TestKNNAlgorithm();
    // TestCUDANNDescent();
    // TestTiledDistanceCompare();
    //TestCUDADistance();
    //TestCUDASearch();
    //TestCUDANewSearch();
    //TestCUDAPriorityQueue();
    //GetRGraph();
}