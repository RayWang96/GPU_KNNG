#include "hip/hip_runtime.h"
#include <assert.h>
#include <hiprand.h>

#include <chrono>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "knncuda_tools.cuh"
#include "knnmerge.cuh"
#include "nndescent.cuh"

#ifdef __INTELLISENSE__
#include "../intellisense_cuda_intrinsics.h"
#endif
using namespace std;

__global__ void CopySecondHalfToKNNGraph(NNDElement *knngraph,
                                         const NNDElement *knngraph_first,
                                         const int knngraph_first_size,
                                         const NNDElement *knngraph_second,
                                         const int knngraph_second_size,
                                         const int *random_knngraph) {
  int list_id = blockIdx.x;
  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  int knngraph_pos_base = list_id * NEIGHB_NUM_PER_LIST;
  int rand_knngraph_pos_base = list_id * LAST_HALF_NEIGHB_NUM;

  if (list_id < knngraph_first_size) {
    if (tx < WARP_SIZE) {
      int it_num = GetItNum(FIRST_HALF_NEIGHB_NUM, WARP_SIZE);
      for (int i = 0; i < it_num; i++) {
        int neighb_pos = i * WARP_SIZE + lane_id;
        if (neighb_pos >= FIRST_HALF_NEIGHB_NUM) break;
        knngraph[knngraph_pos_base + neighb_pos] =
            knngraph_first[knngraph_pos_base + neighb_pos];
      }
    } else {
      int it_num = GetItNum(LAST_HALF_NEIGHB_NUM, WARP_SIZE);
      for (int i = 0; i < it_num; i++) {
        int neighb_pos = i * WARP_SIZE + lane_id;
        if (neighb_pos >= LAST_HALF_NEIGHB_NUM) break;
        auto &elem =
            knngraph[knngraph_pos_base + FIRST_HALF_NEIGHB_NUM + neighb_pos];
        elem.SetDistance(1e10);
        elem.SetLabel(random_knngraph[rand_knngraph_pos_base + neighb_pos] +
                      knngraph_first_size);
      }
    }
  } else {
    int knngraph_second_pos_base =
        (list_id - knngraph_first_size) * NEIGHB_NUM_PER_LIST;
    rand_knngraph_pos_base =
        (list_id - knngraph_first_size) * LAST_HALF_NEIGHB_NUM;
    if (tx < WARP_SIZE) {
      int it_num = GetItNum(FIRST_HALF_NEIGHB_NUM, WARP_SIZE);
      for (int i = 0; i < it_num; i++) {
        int neighb_pos = i * WARP_SIZE + lane_id;
        if (neighb_pos >= FIRST_HALF_NEIGHB_NUM) break;
        auto elem = knngraph_second[knngraph_second_pos_base + neighb_pos];
        elem.SetLabel(elem.label() + knngraph_first_size);
        knngraph[knngraph_pos_base + neighb_pos] = elem;
      }
    } else {
      int it_num = GetItNum(LAST_HALF_NEIGHB_NUM, WARP_SIZE);
      for (int i = 0; i < it_num; i++) {
        int neighb_pos = i * WARP_SIZE + lane_id;
        if (neighb_pos >= LAST_HALF_NEIGHB_NUM) break;
        auto &elem =
            knngraph[knngraph_pos_base + FIRST_HALF_NEIGHB_NUM + neighb_pos];
        elem.SetDistance(1e10);
        elem.SetLabel(random_knngraph[rand_knngraph_pos_base + neighb_pos]);
      }
    }
  }
}

__global__ void InitRandomBlockedKNNGraph(NNDElement *knngraph,
                                          const NNDElement *knngraph_first,
                                          const int knngraph_first_size,
                                          const NNDElement *knngraph_second,
                                          const int knngraph_second_size) {
  __shared__ NNDElement knnlist_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int blocks_size[NEIGHB_BLOCKS_NUM];
  __shared__ int current_block_id;
  int list_id = blockIdx.x;
  int global_pos_base = list_id * NEIGHB_NUM_PER_LIST;
  int merged_size = knngraph_first_size + knngraph_second_size;
  int tx = threadIdx.x;
  if (tx < NEIGHB_BLOCKS_NUM) {
    blocks_size[tx] = 0;
  }

  if (list_id < knngraph_first_size) {
    int it_num = GetItNum(FIRST_HALF_NEIGHB_NUM, WARP_SIZE);
    for (int i = 0; i < it_num; i++) {
      int pos = i * WARP_SIZE + tx;
      if (pos < FIRST_HALF_NEIGHB_NUM) {
        NNDElement elem = knngraph_first[global_pos_base + pos];
        int block_id = elem.label() % NEIGHB_BLOCKS_NUM;
        int new_pos = atomicAdd(&blocks_size[block_id], 1);
        if (new_pos >= WARP_SIZE) {
          atomicExch(&blocks_size[block_id], WARP_SIZE);
        } else {
          knnlist_cache[block_id * WARP_SIZE + new_pos] = elem;
        }
      }
    }
  } else {
    int knngraph_second_pos_base =
        (list_id - knngraph_first_size) * NEIGHB_NUM_PER_LIST;
    int it_num = GetItNum(FIRST_HALF_NEIGHB_NUM, WARP_SIZE);
    for (int i = 0; i < it_num; i++) {
      int pos = i * WARP_SIZE + tx;
      if (pos < FIRST_HALF_NEIGHB_NUM) {
        NNDElement elem = knngraph_second[knngraph_second_pos_base + pos];
        elem.SetLabel(elem.label() + knngraph_first_size);
        elem.MarkOld();
        int block_id = elem.label() % NEIGHB_BLOCKS_NUM;
        int new_pos = atomicAdd(&blocks_size[block_id], 1);
        if (new_pos >= WARP_SIZE) {
          atomicExch(&blocks_size[block_id], WARP_SIZE);
        } else {
          knnlist_cache[block_id * WARP_SIZE + new_pos] = elem;
        }
      }
    }
  }
  if (tx == 0) {
    current_block_id = 0;
  }
  int used_num = 0;
  for (int i = 0; i < NEIGHB_BLOCKS_NUM; i++) {
    int it_num = GetItNum(LAST_HALF_NEIGHB_NUM - used_num, WARP_SIZE);
    int tmp_used_num = used_num + (WARP_SIZE - blocks_size[i]);
    for (int j = 0; j < it_num; j++) {
      int pos = used_num + j * WARP_SIZE + tx;
      if (pos >= LAST_HALF_NEIGHB_NUM) break;
      int new_pos = atomicAdd(&blocks_size[i], 1);
      if (new_pos >= WARP_SIZE) {
        atomicExch(&blocks_size[i], WARP_SIZE);
        break;
      }
      NNDElement elem(1e10, 12345678);
      int new_label;
      if (list_id < knngraph_first_size) {
        int rand_knngraph_pos_base = list_id * LAST_HALF_NEIGHB_NUM;
        new_label =
            xorshift64star(rand_knngraph_pos_base) % knngraph_second_size;
        while (new_label % NEIGHB_BLOCKS_NUM != i || new_label == list_id) {
          new_label = xorshift64star(new_label) % knngraph_second_size;
        }
      } else {
        int rand_knngraph_pos_base =
            (list_id - knngraph_first_size) * LAST_HALF_NEIGHB_NUM;
        new_label =
            xorshift64star(rand_knngraph_pos_base) % knngraph_first_size;
        while (new_label % NEIGHB_BLOCKS_NUM != i || new_label == list_id) {
          new_label = xorshift64star(new_label) % knngraph_first_size;
        }
      }
      elem.SetLabel(new_label);
      knnlist_cache[i * WARP_SIZE + new_pos] = elem;
    }
    used_num = tmp_used_num;
  }
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int pos = i * WARP_SIZE + tx;
    if (pos < NEIGHB_NUM_PER_LIST)
      knngraph[global_pos_base + pos] = knnlist_cache[pos];
  }
}

void PrepareGraphForMerge(NNDElement **knngraph_dev_ptr,
                          NNDElement *knngraph_first_dev,
                          const int knngraph_first_size,
                          NNDElement *knngraph_second_dev,
                          const int knngraph_second_size,
                          const bool free_subgraph = false) {
  NNDElement *&knngraph_dev = *knngraph_dev_ptr;
  int merged_graph_size = knngraph_first_size + knngraph_second_size;
  hipMalloc(&knngraph_dev, (size_t)merged_graph_size * NEIGHB_NUM_PER_LIST *
                                sizeof(NNDElement));
  // CopySecondHalfToKNNGraph<<<merged_graph_size, WARP_SIZE * 2>>>(
  //     knngraph_dev, knngraph_first_dev, knngraph_first_size,
  //     knngraph_second_dev, knngraph_second_size, random_knngraph_dev);
  InitRandomBlockedKNNGraph<<<merged_graph_size, WARP_SIZE>>>(
      knngraph_dev, knngraph_first_dev, knngraph_first_size,
      knngraph_second_dev, knngraph_second_size);
  hipDeviceSynchronize();
  // vector<vector<NNDElement>> g;
  // ToHostKNNGraph(&g, knngraph_dev, merged_graph_size, NEIGHB_NUM_PER_LIST);
  // OutputHostKNNGraph(g, "/home/hwang/codes/GPU_KNNG/results/tmpg.txt");
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    exit(-1);
  }
  if (free_subgraph) {
    hipFree(knngraph_first_dev);
    hipFree(knngraph_second_dev);
  }
}

void MergeVectors(float **vectors_dev_ptr, float *vectors_first_dev,
                  const int vectors_first_size, float *vectors_second_dev,
                  const int vectors_second_size,
                  const bool free_sub_data = false) {
  float *&vectors_dev = *vectors_dev_ptr;
  int merged_size = vectors_first_size + vectors_second_size;
  hipMalloc(&vectors_dev, (size_t)merged_size * VEC_DIM * sizeof(float));
  hipMemcpyAsync(vectors_dev, vectors_first_dev,
                  (size_t)vectors_first_size * VEC_DIM * sizeof(float),
                  hipMemcpyDeviceToDevice);
  if (free_sub_data) {
    hipFree(vectors_first_dev);
  }
  hipMemcpyAsync(vectors_dev + (size_t)vectors_first_size * VEC_DIM,
                  vectors_second_dev,
                  (size_t)vectors_second_size * VEC_DIM * sizeof(float),
                  hipMemcpyDeviceToDevice);
  if (free_sub_data) {
    hipFree(vectors_second_dev);
  }
}

namespace gpuknn {
void KNNMerge(NNDElement **knngraph_merged_dev_ptr, float *vectors_first_dev,
              const int vectors_first_size, NNDElement *knngraph_first_dev,
              float *vectors_second_dev, const int vectors_second_size,
              NNDElement *knngraph_second_dev, const bool free_sub_data) {
  NNDElement *&knngraph_merged_dev = *knngraph_merged_dev_ptr;
  float *vectors_dev;
  int merged_graph_size = vectors_first_size + vectors_second_size;
  auto start = chrono::steady_clock::now();
  MarkAllToOld<<<vectors_first_size, NEIGHB_NUM_PER_LIST>>>(knngraph_first_dev);
  MarkAllToOld<<<vectors_second_size, NEIGHB_NUM_PER_LIST>>>(
      knngraph_second_dev);
  hipDeviceSynchronize();
  PrepareGraphForMerge(&knngraph_merged_dev, knngraph_first_dev,
                       vectors_first_size, knngraph_second_dev,
                       vectors_second_size, free_sub_data);
  MergeVectors(&vectors_dev, vectors_first_dev, vectors_first_size,
               vectors_second_dev, vectors_second_size, free_sub_data);
  auto end = chrono::steady_clock::now();
  float time_cost =
      (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() /
      1e6;
  cerr << "PrepareGraphForMerge costs: " << time_cost << endl;
  NNDescentRefine(knngraph_merged_dev, vectors_dev, merged_graph_size, VEC_DIM,
                  3);
  hipFree(vectors_dev);
}

void KNNMergeFromHost(NNDElement **knngraph_merged_dev_ptr,
                      const float *vectors_first, const int vectors_first_size,
                      const NNDElement *knngraph_first,
                      const float *vectors_second,
                      const int vectors_second_size,
                      const NNDElement *knngraph_second) {
  NNDElement *&knngraph_merged_dev = *knngraph_merged_dev_ptr;
  float *vectors_dev;
  int merged_graph_size = vectors_first_size + vectors_second_size;
  auto start = chrono::steady_clock::now();
  NNDElement *knngraph_first_dev, *knngraph_second_dev;
  hipMalloc(&knngraph_first_dev, (size_t)vectors_first_size *
                                      NEIGHB_NUM_PER_LIST * sizeof(NNDElement));
  hipMalloc(
      &knngraph_second_dev,
      (size_t)vectors_second_size * NEIGHB_NUM_PER_LIST * sizeof(NNDElement));
  hipMemcpy(
      knngraph_first_dev, knngraph_first,
      (size_t)vectors_first_size * NEIGHB_NUM_PER_LIST * sizeof(NNDElement),
      hipMemcpyHostToDevice);
  hipMemcpy(
      knngraph_second_dev, knngraph_second,
      (size_t)vectors_second_size * NEIGHB_NUM_PER_LIST * sizeof(NNDElement),
      hipMemcpyHostToDevice);

  MarkAllToOld<<<vectors_first_size, NEIGHB_NUM_PER_LIST>>>(knngraph_first_dev);
  MarkAllToOld<<<vectors_second_size, NEIGHB_NUM_PER_LIST>>>(
      knngraph_second_dev);
  hipDeviceSynchronize();
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << "Mark failed " << hipGetErrorString(cuda_status) << endl;
    exit(-1);
  }

  // Dev. ptrs are freed inside the function.
  PrepareGraphForMerge(&knngraph_merged_dev, knngraph_first_dev,
                       vectors_first_size, knngraph_second_dev,
                       vectors_second_size, true);

  float *vectors_first_dev, *vectors_second_dev;
  hipMalloc(&vectors_first_dev,
             (size_t)vectors_first_size * VEC_DIM * sizeof(float));
  hipMalloc(&vectors_second_dev,
             (size_t)vectors_second_size * VEC_DIM * sizeof(float));
  hipMemcpy(vectors_first_dev, vectors_first,
             (size_t)vectors_first_size * VEC_DIM * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(vectors_second_dev, vectors_second,
             (size_t)vectors_second_size * VEC_DIM * sizeof(float),
             hipMemcpyHostToDevice);

  // Dev. ptrs are freed inside the function.
  MergeVectors(&vectors_dev, vectors_first_dev, vectors_first_size,
               vectors_second_dev, vectors_second_size, true);
  auto end = chrono::steady_clock::now();
  float time_cost =
      (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() /
      1e6;
  cerr << "PrepareGraphForMerge costs: " << time_cost << endl;

  NNDescentRefine(knngraph_merged_dev, vectors_dev, merged_graph_size, VEC_DIM,
                  5);
  hipFree(vectors_dev);
}
}  // namespace gpuknn