#include "hip/hip_runtime.h"
#ifndef XMUKNN_NNDESCENT_CU
#define XMUKNN_NNDESCENT_CU

#include <vector>
#include <iostream>
#include <assert.h>
#include <bitset>
#include <algorithm>
#include <cstring>
#include <tuple>
#include <utility>
#include <chrono>
#include <mutex> 
#include <mma.h>

#include "result_element.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "nndescent.cuh"
#include "../xmuknn.h"
#include "../tools/distfunc.hpp"

#ifdef __INTELLISENSE__
#include "../intellisense_cuda_intrinsics.h"
#endif

using namespace std;
using namespace xmuknn;
#define DEVICE_ID 0
#define LARGE_INT 0x3f3f3f3f
const int VEC_DIM = 128;
const int NEIGHB_NUM_PER_LIST = 40;
const int NEIGHB_CACHE_NUM = 16;
const int TILE_WIDTH = 16;
const int THREADS_PER_LIST = 32;
const int SAMPLE_NUM = 30;
__device__ int for_check = 0;

void GetNBGraph(Graph *graph_new_ptr,
                Graph *graph_old_ptr,
                vector<vector<gpuknn::NNDItem>>& knn_graph, 
                const float *vectors, const int vecs_size, 
                const int vecs_dim) {
    auto time1 = chrono::steady_clock::now();
    int sample_num = SAMPLE_NUM;
    Graph &graph_new = *graph_new_ptr;
    Graph &graph_old = *graph_old_ptr;
    Graph graph_rnew, graph_rold;
    graph_new = graph_rnew = graph_old = graph_rold = Graph(knn_graph.size());
    vector<mutex> mtx(vecs_size);

    #pragma omp parallel for
    for (int i = 0; i < knn_graph.size(); i++) {
        int cnt = 0;
        for (int j = 0; j < knn_graph[i].size(); j++) {
            auto& item = knn_graph[i][j];
            if (item.visited) {
                graph_old[i].push_back(item.id);
            }
            else {
                if (cnt < sample_num) {
                    graph_new[i].push_back(item.id);
                    cnt++;
                    item.visited = true;
                }
            }
            if (cnt >= sample_num) break;
        }
    }
    // auto time2 = chrono::steady_clock::now();
    // cerr << "Mark 2: " << (float)chrono::duration_cast<std::chrono::microseconds>(time2 - time1).count() / 1e6 << endl;

    #pragma omp parallel for
    for (int i = 0; i < knn_graph.size(); i++) {
        for (int j = 0; j < graph_new[i].size(); j++) {
            auto& id = graph_new[i][j];
            mtx[id].lock();
            graph_rnew[id].push_back(i);
            mtx[id].unlock();
        }
        for (int j = 0; j < graph_old[i].size(); j++) {
            auto& id = graph_old[i][j];
            mtx[id].lock();
            graph_rold[id].push_back(i);
            mtx[id].unlock();
        }
    }

    // auto time3 = chrono::steady_clock::now();
    // cerr << "Mark 3: " << (float)chrono::duration_cast<std::chrono::microseconds>(time3 - time1).count() / 1e6 << endl;

    // #pragma omp parallel for
    // for (int i = 0; i < knn_graph.size(); i++) {
    //     random_shuffle(graph_rnew[i].begin(), graph_rnew[i].end());
    //     random_shuffle(graph_rold[i].begin(), graph_rold[i].end());
    // }

    // auto time4 = chrono::steady_clock::now();
    // cerr << "Mark 4: " << (float)chrono::duration_cast<std::chrono::microseconds>(time4 - time1).count() / 1e6 << endl;

    vector<bool> visited(vecs_size);
    // #pragma omp parallel for
    for (int i = 0; i < knn_graph.size(); i++) {
        int cnt = 0;
        for (int j = 0; j < graph_new[i].size(); j++) {
            visited[graph_new[i][j]] = true;
        }
        for (int j = 0; j < graph_old[i].size(); j++) {
            visited[graph_old[i][j]] = true;
        }
        for (int j = 0; j < graph_rnew[i].size(); j++) {
            int x = graph_rnew[i][j];
            if (!visited[x]) {
                cnt++;
                visited[x] = true;
                graph_new[i].push_back(x);
                if (cnt >= sample_num) break;
            }
        }
        cnt = 0;
        for (int j = 0; j < graph_rold[i].size(); j++) {
            int x = graph_rold[i][j];
            if (!visited[x]) {
                cnt++;
                visited[x] = true;
                graph_old[i].push_back(x);
                if (cnt >= sample_num) break;
            }
        }
        for (int j = 0; j < graph_new[i].size(); j++) {
            visited[graph_new[i][j]] = false;
        }
        for (int j = 0; j < graph_old[i].size(); j++) {
            visited[graph_old[i][j]] = false;
        }
    }

    
    // auto time5 = chrono::steady_clock::now();
    // cerr << "Mark 5: " << (float)chrono::duration_cast<std::chrono::microseconds>(time5 - time1).count() / 1e6 << endl;

    // #pragma omp parallel for
    for (int i = 0; i < knn_graph.size(); i++) {
        sort(graph_new[i].begin(), graph_new[i].end());
        graph_new[i].erase(unique(graph_new[i].begin(), 
                                  graph_new[i].end()), graph_new[i].end());

        sort(graph_old[i].begin(), graph_old[i].end());
        graph_old[i].erase(unique(graph_old[i].begin(), 
                                  graph_old[i].end()), graph_old[i].end());
    }
    // auto time6 = chrono::steady_clock::now();
    // cerr << "Mark 6: " << (float)chrono::duration_cast<std::chrono::microseconds>(time6 - time1).count() / 1e6 << endl;
    return;
}

__device__ int GetItNum(const int sum_num, const int num_per_it) {
    return sum_num / num_per_it + (sum_num % num_per_it != 0);
}

__device__ void Swap(int &a, int &b) {
    int c = a;
    a = b;
    b = c;
}

__device__ __forceinline__ ResultElement XorSwap(ResultElement x, int mask, int dir) {
    ResultElement y;
    y.distance = __shfl_xor_sync(0xffffffff, x.distance, mask, THREADS_PER_LIST);
    y.label = __shfl_xor_sync(0xffffffff, x.label, mask, THREADS_PER_LIST);
    return x < y == dir ? y : x;
}

__device__ __forceinline__ int Bfe(int lane_id, int pos) {
    int res;
    asm("bfe.u32 %0,%1,%2,%3;"
        : "=r"(res) : "r"(lane_id), "r"(pos),"r"(1));
    return res;
}

__device__ void BitonicSort(ResultElement *sort_element_ptr, const int lane_id) {
    auto &sort_elem = *sort_element_ptr;
    sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 1) ^ Bfe(lane_id, 0));
    sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 2) ^ Bfe(lane_id, 1));
    sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 2) ^ Bfe(lane_id, 0));
    sort_elem = XorSwap(sort_elem, 0x04, Bfe(lane_id, 3) ^ Bfe(lane_id, 2));
    sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 3) ^ Bfe(lane_id, 1));
    sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 3) ^ Bfe(lane_id, 0));
    sort_elem = XorSwap(sort_elem, 0x08, Bfe(lane_id, 4) ^ Bfe(lane_id, 3));
    sort_elem = XorSwap(sort_elem, 0x04, Bfe(lane_id, 4) ^ Bfe(lane_id, 2));
    sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 4) ^ Bfe(lane_id, 1));
    sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 4) ^ Bfe(lane_id, 0));
    sort_elem = XorSwap(sort_elem, 0x10,                   Bfe(lane_id, 4));
    sort_elem = XorSwap(sort_elem, 0x08,                   Bfe(lane_id, 3));
    sort_elem = XorSwap(sort_elem, 0x04,                   Bfe(lane_id, 2));
    sort_elem = XorSwap(sort_elem, 0x02,                   Bfe(lane_id, 1));
    sort_elem = XorSwap(sort_elem, 0x01,                   Bfe(lane_id, 0));
    return;
}

__device__ void UpdateLocalKNNLists(ResultElement *knn_list,
                                    const int *neighbs_id,
                                    const int list_id,
                                    const int list_size,
                                    const float *distances,
                                    const int distances_num) {
    int head_pos = list_id * (list_id - 1) / 2;
    int tail_pos = (list_id + 1) * list_id / 2;
    int y_num = tail_pos - head_pos;

    int tx = threadIdx.x;
    int lane_id = tx % THREADS_PER_LIST;
    int pos_in_lists = list_id * NEIGHB_CACHE_NUM;

    int it_num = GetItNum(y_num, THREADS_PER_LIST);
    for (int it = 0; it < it_num; it++) {
        // bitonic sort
        ResultElement sort_elem;
        sort_elem.label = neighbs_id[it * THREADS_PER_LIST + lane_id];
        int current_pos = head_pos + it * THREADS_PER_LIST + lane_id;
        if (current_pos < tail_pos) {
            sort_elem.distance = distances[current_pos];
        } else {
            sort_elem.distance = 1e10;
            sort_elem.label = 87654321;
        }
        // printf("%d %f %d\n", lane_id, sort_elem.distance, sort_elem.label);
        BitonicSort(&sort_elem, lane_id);
        int offset;
        for (offset = 0; offset < NEIGHB_CACHE_NUM; offset++) {
            int flag = 1;
            if (lane_id == THREADS_PER_LIST - 1) {
                if (sort_elem < knn_list[pos_in_lists + offset]) {
                    flag = 0;
                }
            }
            flag = __shfl_sync(0xffffffff, flag, THREADS_PER_LIST - 1, 
                               THREADS_PER_LIST);
            if (!flag) break;
            ResultElement tmp;
            tmp.distance = __shfl_up_sync(0xffffffff, sort_elem.distance, 
                                          1, THREADS_PER_LIST);
            tmp.label = __shfl_up_sync(0xffffffff, sort_elem.label, 
                                       1, THREADS_PER_LIST);
            sort_elem = tmp;
        }
        if (lane_id < offset) {
            if (lane_id < NEIGHB_CACHE_NUM) {
                sort_elem = knn_list[pos_in_lists + lane_id];
            } else {
                sort_elem = ResultElement(1e10, 12345678);
            }        
        }
        BitonicSort(&sort_elem, lane_id);
        if (lane_id < NEIGHB_CACHE_NUM)
            knn_list[pos_in_lists + lane_id] = sort_elem;
    }

    head_pos = list_id * (list_id + 3) / 2; // 0   2   5   9   14
    for (int it = 0; it < 2; it++) {
        ResultElement sort_elem;
        int no = it * THREADS_PER_LIST + lane_id;
        sort_elem.label = neighbs_id[no + list_id + 1];
        int current_pos = head_pos + no * (no + list_id * 2 + 1) / 2;
        if (current_pos < distances_num) {
            sort_elem.distance = distances[current_pos];
        } else {
            sort_elem.distance = 1e10;
            sort_elem.label = 99999999;
        }
        BitonicSort(&sort_elem, lane_id);
        int offset;
        for (offset = 0; offset < NEIGHB_CACHE_NUM; offset++) {
            int flag = 1;
            if (lane_id == THREADS_PER_LIST - 1) {
                if (sort_elem < knn_list[pos_in_lists + offset]) {
                    flag = 0;
                }
            }
            flag = __shfl_sync(0xffffffff, flag, THREADS_PER_LIST - 1, 
                               THREADS_PER_LIST);
            if (!flag) break;
            ResultElement tmp;
            tmp.distance = __shfl_up_sync(0xffffffff, sort_elem.distance, 
                                          1, THREADS_PER_LIST);
            tmp.label = __shfl_up_sync(0xffffffff, sort_elem.label, 
                                       1, THREADS_PER_LIST);
            sort_elem = tmp;
        }
        if (lane_id < offset) {
            if (lane_id < NEIGHB_CACHE_NUM) {
                sort_elem = knn_list[pos_in_lists + lane_id];
            } else {
                sort_elem = ResultElement(1e10, 12345678);
            }
        }
        BitonicSort(&sort_elem, lane_id);
        if (lane_id < NEIGHB_CACHE_NUM)
            knn_list[pos_in_lists + lane_id] = sort_elem;
    }

    // printf("%d %f %d\n", lane_id, knn_list[pos_in_lists + lane_id].distance, knn_list[pos_in_lists + lane_id].label);
}

__device__ void UpdateLocalNewKNNLists(ResultElement *knn_list,
                                       const int list_id,
                                       const int list_size,
                                       const int *old_neighbs,
                                       const int num_old,
                                       const float *distances,
                                       const int distances_num) {
    int head_pos = list_id * num_old;
    int y_num = num_old;
    int tail_pos = head_pos + num_old;

    int tx = threadIdx.x;
    int lane_id = tx % THREADS_PER_LIST;
    int pos_in_lists = list_id * NEIGHB_CACHE_NUM;

    int it_num = GetItNum(y_num, THREADS_PER_LIST);
    for (int it = 0; it < it_num; it++) {
        // bitonic sort
        ResultElement sort_elem;
        int no = it * THREADS_PER_LIST + lane_id;
        sort_elem.label = old_neighbs[no];
        int current_pos = head_pos + no;
        if (current_pos < tail_pos) {
            sort_elem.distance = distances[current_pos];
        } else {
            sort_elem.distance = 1e10;
            sort_elem.label = 87654321;
        }
        // printf("%d %f %d\n", lane_id, sort_elem.distance, sort_elem.label);
        BitonicSort(&sort_elem, lane_id);
        int offset;
        for (offset = 0; offset < NEIGHB_CACHE_NUM; offset++) {
            int flag = 1;
            if (lane_id == THREADS_PER_LIST - 1) {
                if (sort_elem < knn_list[pos_in_lists + offset]) {
                    flag = 0;
                }
            }
            flag = __shfl_sync(0xffffffff, flag, THREADS_PER_LIST - 1, 
                               THREADS_PER_LIST);
            if (!flag) break;
            ResultElement tmp;
            tmp.distance = __shfl_up_sync(0xffffffff, sort_elem.distance, 
                                          1, THREADS_PER_LIST);
            tmp.label = __shfl_up_sync(0xffffffff, sort_elem.label, 
                                       1, THREADS_PER_LIST);
            sort_elem = tmp;
        }
        if (lane_id < offset) {
            if (lane_id < NEIGHB_CACHE_NUM) {
                sort_elem = knn_list[pos_in_lists + lane_id];
            } else {
                sort_elem = ResultElement(1e10, 12345678);
            }
        }
        BitonicSort(&sort_elem, lane_id);
        if (lane_id < NEIGHB_CACHE_NUM)
            knn_list[pos_in_lists + lane_id] = sort_elem;
    }
}

__device__ void UpdateLocalOldKNNLists(ResultElement *knn_list,
                                       const int list_id,
                                       const int list_size,
                                       const int *new_neighbs,
                                       const int num_new,
                                       const int *old_neighbs,
                                       const int num_old,
                                       const float *distances,
                                       const int distances_num,
                                       const float *vectors) {
    int head_pos = list_id - num_new;
    int tx = threadIdx.x;
    int lane_id = tx % THREADS_PER_LIST;
    int pos_in_lists = list_id * NEIGHB_CACHE_NUM;

    int it_num = GetItNum(num_new, THREADS_PER_LIST);
    for (int it = 0; it < it_num; it++) {
        ResultElement sort_elem;
        int no = it * THREADS_PER_LIST + lane_id;
        sort_elem.label = new_neighbs[no];
        int current_pos = head_pos + no * num_old;
        if (current_pos < distances_num) {
            sort_elem.distance = distances[current_pos];
        } else {
            sort_elem.distance = 1e10;
            sort_elem.label = 55555555;
        }
        BitonicSort(&sort_elem, lane_id);
        int offset;
        for (offset = 0; offset < NEIGHB_CACHE_NUM; offset++) {
            int flag = 1;
            if (lane_id == THREADS_PER_LIST - 1) {
                if (sort_elem < knn_list[pos_in_lists + offset]) {
                    flag = 0;
                }
            }
            flag = __shfl_sync(0xffffffff, flag, THREADS_PER_LIST - 1, 
                               THREADS_PER_LIST);
            if (!flag) break;
            ResultElement tmp;
            tmp.distance = __shfl_up_sync(0xffffffff, sort_elem.distance, 
                                          1, THREADS_PER_LIST);
            tmp.label = __shfl_up_sync(0xffffffff, sort_elem.label, 
                                       1, THREADS_PER_LIST);
            sort_elem = tmp;
        }
        if (lane_id < offset) {
            if (lane_id < NEIGHB_CACHE_NUM) {
                sort_elem = knn_list[pos_in_lists + lane_id];
            } else {
                sort_elem = ResultElement(1e10, 12345678);
            }
        }
        BitonicSort(&sort_elem, lane_id);
        if (lane_id < NEIGHB_CACHE_NUM)
            knn_list[pos_in_lists + lane_id] = sort_elem;
    }
}

__device__ int InsertToLocalKNNList(ResultElement *knn_list, 
                                    const int list_size,
                                    const ResultElement &element,
                                    int *local_lock_ptr) {
    int &local_lock = *local_lock_ptr;
    int pos = -1;
    bool loop_flag = false;
    do {
        if (loop_flag = atomicCAS(&local_lock, 0, 1) == 0) {
            if (element >= knn_list[list_size-1]) ;
            else {
                int i = 0;
                while (i < list_size && knn_list[i] < element) {
                    i++;
                }
                if (knn_list[i] != element) {
                    for (int j = list_size - 1; j > i && j > 0; j--) {
                        knn_list[j] = knn_list[j-1];
                    }
                    knn_list[i] = element;
                    pos = i;
                }
            }
        }
        __threadfence();
        if (loop_flag) atomicExch(&local_lock, 0);
    } while (!loop_flag);
    return pos;
}

__device__ void UniqueMergeSequential(const ResultElement* A, const int m,
                                      const ResultElement* B, const int n,
                                      ResultElement* C, const int k) {
    int i = 0, j = 0, cnt = 0;
    while ((i < m) && (j < n)) {
        if (A[i] <= B[j]) {
            C[cnt++] = A[i++];
            if (cnt >= k) goto EXIT;
            while (i < m && A[i] <= C[cnt-1]) i++;
            while (j < n && B[j] <= C[cnt-1]) j++;
        } else {
            C[cnt++] = B[j++];
            if (cnt >= k) goto EXIT;
            while (i < m && A[i] <= C[cnt-1]) i++;
            while (j < n && B[j] <= C[cnt-1]) j++;
        }
    }

    if (i == m) {
        for (; j < n; j++) {
            if (B[j] > C[cnt-1]) {
                C[cnt++] = B[j];
            }
            if (cnt >= k) goto EXIT;
        }
        for (; i < m; i++) {
            if (A[i] > C[cnt-1]) {
                C[cnt++] = A[i];
            }
            if (cnt >= k) goto EXIT;
        }
    } else {
        for (; i < m; i++) {
            if (A[i] > C[cnt-1]) {
                C[cnt++] = A[i];
            }
            if (cnt >= k) goto EXIT;
        }
        for (; j < n; j++) {
            if (B[j] > C[cnt-1]) {
                C[cnt++] = B[j];
            }
            if (cnt >= k) goto EXIT;
        }
    }

EXIT:
    if (cnt != k) {
        int flag = atomicCAS(&for_check, 0, 1);
        if (!flag) {
            printf("%d %d\n", cnt, k);
            for (int i = 0; i < m; i++) {
                printf("%f ", A[i].distance);
            } printf("\n\n");
            for (int i = 0; i < n; i++) {
                printf("%f ", B[i].distance);
            } printf("\n\n");
            for (int i = 0; i < k; i++) {
                printf("%f ", C[i].distance);
            } printf("\n\n");

            for (int i = 0; i < m; i++) {
                printf("%d ", A[i].label);
            } printf("\n\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", B[i].label);
            } printf("\n\n");
            for (int i = 0; i < k; i++) {
                printf("%d ", C[i].label);
            } printf("\n\n");
            printf("%d %d\n", cnt, k);
            assert(cnt == k);
        }
    }
    return;
}

__device__ void MergeLocalGraphWithGlobalGraph(const ResultElement* local_knn_graph,
                                               const int list_size, const int* neighb_ids,
                                               const int neighb_num,
                                               ResultElement* global_knn_graph,
                                               int* global_locks) {
    int tx = threadIdx.x;
    if (tx < neighb_num) {
        ResultElement C_cache[NEIGHB_NUM_PER_LIST];
        int neighb_id = neighb_ids[tx];
        bool loop_flag = false;
        do {
            __nanosleep(8);
            if (loop_flag = atomicCAS(&global_locks[neighb_id], 0, 1) == 0) {
                UniqueMergeSequential(&local_knn_graph[tx * NEIGHB_CACHE_NUM], 
                                      NEIGHB_CACHE_NUM, 
                                      &global_knn_graph[neighb_id * NEIGHB_NUM_PER_LIST],
                                      NEIGHB_NUM_PER_LIST, C_cache, NEIGHB_NUM_PER_LIST);
                for (int i = 0; i < NEIGHB_NUM_PER_LIST; i++) {
                    global_knn_graph[neighb_id * NEIGHB_NUM_PER_LIST + i]
                        = C_cache[i];
                }
            }
            __threadfence();
            if (loop_flag) atomicExch(&global_locks[neighb_id], 0);
        } while (!loop_flag);
    }
}

__global__ void NewNeighborsCompareKernel(ResultElement *knn_graph, int *global_locks,
                                          const float *vectors,
                                          const int *edges_new, const int *dest_new,
                                          const int num_new_max) {
    extern __shared__ char buffer[];

    __shared__ float *shared_vectors, *distances;
    __shared__ int *neighbors, *local_locks;
    __shared__ ResultElement *knn_graph_cache;
    __shared__ int pos_gnew, num_new;

    int tx = threadIdx.x;
    if (tx == 0) {
        shared_vectors = (float *)buffer;
        distances = 
            (float *)((char *)buffer + num_new_max * VEC_DIM * sizeof(float));
        neighbors = 
            (int *)((char *)distances + (num_new_max * (num_new_max - 1) / 2) * sizeof(float));
        local_locks = (int *)((char *)neighbors + num_new_max * sizeof(int));
        knn_graph_cache = 
            (ResultElement *)((char *)local_locks + num_new_max * sizeof(int));
    }
    __syncthreads();

    int list_id = blockIdx.x;
    int block_dim_x = blockDim.x;

    if (tx < num_new_max) {
        local_locks[tx] = 0;
    }

    if (tx == 0) {
        int next_pos = edges_new[list_id + 1];
        int now_pos = edges_new[list_id];
        num_new = next_pos - now_pos;
        pos_gnew = now_pos;
    }
    __syncthreads();
    int neighb_num = num_new;
    if (tx < neighb_num) {
        neighbors[tx] = dest_new[pos_gnew + tx];
    }
    __syncthreads();
    int num_vec_per_it = block_dim_x / VEC_DIM;
    int num_it = GetItNum(neighb_num, num_vec_per_it);
    for (int i = 0; i < num_it; i++) {
        int x = i * num_vec_per_it + tx / VEC_DIM;
        if (x >= neighb_num) continue;
        int y = tx % VEC_DIM;
        int vec_id = neighbors[x];
        shared_vectors[x * VEC_DIM + y] = vectors[vec_id * VEC_DIM + y];
    }

    int calc_num = (neighb_num * (neighb_num - 1)) / 2;

    num_it = GetItNum(calc_num, block_dim_x);
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("check calc. num. %d %d\n", neighb_num, calc_num);
    }
    for (int i = 0; i < num_it; i++) {
        int x = i * block_dim_x + tx;
        if (x < calc_num) {
            distances[x] = 0;
        }
    }
    __syncthreads();
    for (int i = 0; i < num_it; i++) {
        int no = i * block_dim_x + tx;
        if (no >= calc_num) continue;
        int idx = no + 1;
        int x = ceil(sqrt(2 * idx + 0.25) - 0.5);
        int y = idx - (x - 1) * x / 2 - 1;
        if (x >= neighb_num || y >= neighb_num) continue;
        float sum = 0;
        for (int j = 0; j < VEC_DIM; j++) {
            float diff = shared_vectors[x * VEC_DIM + j] - 
                         shared_vectors[y * VEC_DIM + j];
            sum += diff * diff;
        }
        distances[no] = sum;
    }
    __syncthreads();
    // num_it = GetItNum(NEIGHB_NUM_PER_LIST, NEIGHB_CACHE_NUM);
    num_it = 1;
    for (int i = 0; i < num_it; i++) {
        int num_it2 = GetItNum(neighb_num * NEIGHB_CACHE_NUM, block_dim_x);
        for (int j = 0; j < num_it2; j++) {
            int pos = j * block_dim_x + tx;
            if (pos < neighb_num * NEIGHB_CACHE_NUM)
                knn_graph_cache[pos] = ResultElement(1e10, 77777777);
        }
        int list_size = NEIGHB_CACHE_NUM;
        int num_it3 = GetItNum(neighb_num, block_dim_x / THREADS_PER_LIST);
        for (int j = 0; j < num_it3; j++) {
            int list_id = j * (block_dim_x / THREADS_PER_LIST) + tx / THREADS_PER_LIST;
            if (list_id >= neighb_num) continue;
            UpdateLocalKNNLists(knn_graph_cache, neighbors, 
                                list_id, list_size, distances, calc_num);
        }
        __syncthreads();
        MergeLocalGraphWithGlobalGraph(knn_graph_cache, NEIGHB_CACHE_NUM, neighbors,
                                       neighb_num, knn_graph, global_locks);
        __syncthreads();
    }
}


// blockDim.x = TILE_WIDTH * TILE_WIDTH;
__device__ void GetNewOldDistances(float *distances, const float *vectors,
                                   const int *new_neighbors, const int num_new,
                                   const int *old_neighbors, const int num_old) {
    __shared__ float nsv[TILE_WIDTH][TILE_WIDTH]; //New shared vectors
    __shared__ float osv[TILE_WIDTH][TILE_WIDTH]; //Old shared vectors
    const int tile_size = TILE_WIDTH * TILE_WIDTH;
    const int width = VEC_DIM;

    int tx = threadIdx.x;
    int t_row = tx / TILE_WIDTH;
    int t_col = tx % TILE_WIDTH;
    int row_num = (int)(ceil(1.0 * num_new / TILE_WIDTH));
    int col_num = (int)(ceil(1.0 * num_old / TILE_WIDTH));
    int tiles_num = row_num * col_num;
    // if (threadIdx.x == 0) {
    //     printf("%d %d %d\n", row_num, col_num, tiles_num);
    // }
    for (int i = 0; i < tiles_num; i++) {
        float distance = -1.0;
        int row_new = i / col_num * TILE_WIDTH;
        int row_old = i % col_num * TILE_WIDTH;

        // Assume that the dimension of vectors larger than num of neighbors.
        for (int ph = 0; ph < ceil(width / (float)TILE_WIDTH); ph++) {
            if ((row_new + t_row < num_new) && (ph * TILE_WIDTH + t_col < VEC_DIM)) {
                nsv[t_row][t_col] = 
                    vectors[new_neighbors[row_new + t_row] * VEC_DIM +
                            ph * TILE_WIDTH + t_col];
            } else {
                nsv[t_row][t_col] = 1e10;
            }

            if ((row_old + t_col < num_old) && (ph * TILE_WIDTH + t_row < VEC_DIM)) {
                osv[t_col][t_row] = 
                    vectors[old_neighbors[row_old + t_col] * VEC_DIM +
                            ph * TILE_WIDTH + t_row];
            } else {
                osv[t_col][t_row] = 1e10;
            }
            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; k++) {
                float a = nsv[t_row][k], b = osv[t_col][k];
                if (a > 1e9 || b > 1e9) {
                }
                else {
                    float diff = a - b;
                    if (distance == -1.0) distance = 0;
                    distance += diff * diff;
                }
            }
            __syncthreads();
        }
        if (distance != -1.0) {
            distances[(row_new + t_row) * num_old + row_old + t_col] = distance;
        }
    }
}

const int M = 16;
const int N = 16;
const int K = 16;

const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

const int BLOCK_COL_WARPS = 1;
const int BLOCK_ROW_WARPS = 4;

const int ROWS_PER_IT = BLOCK_ROW_WARPS * WMMA_M;
const int COLS_PER_IT = BLOCK_COL_WARPS * WMMA_K;

using namespace nvcuda;
// threads = 4 * 32 = 128
__device__ void GetDistancesWMMA(float *distances, 
                                 const float *vectors,
                                 const int *new_neighbors, const int num_new,
                                 const int *old_neighbors, const int num_old,
                                 half shmem_a[][WMMA_K], 
                                 half shmem_b[][WMMA_K]) {
    __shared__ float distances_cache[4 * 4 * 16 * 16];
    __shared__ float squa_suma_cache[64];
    __shared__ float squa_sumb_cache[64];

    const int tx = threadIdx.x;
    const int warp_id = tx / warpSize;
    const int lane_id = tx % warpSize;
    int arow_it = GetItNum(num_new, ROWS_PER_IT);
    int col_it = GetItNum(VEC_DIM, COLS_PER_IT);
    int brow_it = GetItNum(num_old, ROWS_PER_IT);
    for (int i = 0; i < arow_it; i++) {
        int local_base_ay = warp_id * WMMA_M;
        int global_base_ay = i * ROWS_PER_IT + local_base_ay;
        for (int j = 0; j < brow_it; j++) {
            int local_base_by = warp_id * WMMA_N;
            int global_base_by = j * ROWS_PER_IT + local_base_by;
            wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::row_major>
                a_frag;
            wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major>
                b_frag;
            wmma::fragment<wmma::accumulator, M, N, K, float> 
                acc_frag[BLOCK_ROW_WARPS];
            for (int k = 0; k < ROWS_PER_IT; k++) {
                wmma::fill_fragment(acc_frag[k], 0.0f);
            } 
            float squa_suma = 0, squa_sumb = 0;
            for (int k = 0; k < col_it; k++) {
                int local_x = lane_id % WMMA_K;
                int global_x = k * COLS_PER_IT + local_x; 
                if (lane_id < 16) {
                    for (int t = 0; t < WMMA_M; t++) {
                        int global_ay = global_base_ay + t;
                        int local_ay = local_base_ay + t;

                        int global_by = global_base_by + t;
                        int local_by = local_base_by + t;
                        if (global_ay < num_new && global_x < VEC_DIM) {
                            int pos = new_neighbors[global_ay];
                            float val = vectors[pos * VEC_DIM + global_x];
                            shmem_a[local_ay][local_x] = (half)val;
                        } else {
                            shmem_a[local_ay][local_x] = (half)0.0;
                        }
                        if (global_by < num_old && global_x < VEC_DIM) {
                            int pos = old_neighbors[global_by];
                            float val = vectors[pos * VEC_DIM + global_x];
                            shmem_b[local_by][local_x] = (half)val;
                        } else {
                            shmem_b[local_by][local_x] = (half)0.0;
                        }
                    }
                } 
                __syncthreads();
                if (lane_id < 16) {
                    for (int t = 0; t < WMMA_K; t++) {
                        float val = (float)shmem_a[warp_id * 16 + lane_id][t];
                        squa_suma += val * val;
                        val = (float)shmem_b[warp_id * 16 + lane_id][t];
                        squa_sumb += val * val;
                    }
                }
                // if (num_old > 0 && k == 0) {
                //     int flag = atomicCAS(&for_check, 0, 1);
                //     if (!flag) {
                //         for (int ii = 0; ii < num_old; ii++) {
                //             for (int jj = 0; jj < 16; jj++) {
                //                 int x = j * ROWS_PER_IT + ii;
                //                 int y = k * COLS_PER_IT + jj;
                //                 printf("%f ", vectors[old_neighbors[x] * VEC_DIM + y]);
                //             } printf("\n");
                //         } printf("\n\n");
                //         for (int i = 0; i < num_old; i++) {
                //             for (int j = 0; j < 16; j++) {
                //                 printf("%f ", (float)shmem_b[i][j]);
                //             } printf("\n");
                //         } printf("\n\n");
                //     }
                // }
                // __syncthreads();
                // if (num_new > 0 && k == 0) {
                //     int flag = atomicCAS(&for_check, 0, 1);
                //     if (!flag) {
                //         for (int ii = 0; ii < num_new; ii++) {
                //             for (int jj = 0; jj < 16; jj++) {
                //                 int x = i * ROWS_PER_IT + ii;
                //                 int y = k * COLS_PER_IT + jj;
                //                 printf("%f ", vectors[new_neighbors[x] * VEC_DIM + y]);
                //             } printf("\n");
                //         } printf("\n\n");
                //         for (int i = 0; i < num_new; i++) {
                //             for (int j = 0; j < 16; j++) {
                //                 printf("%f ", (float)shmem_a[i][j]);
                //             }printf("\n");
                //         } printf("\n\n");
                //     }
                // }
                // __syncthreads();
                wmma::load_matrix_sync(a_frag, 
                                       &shmem_a[warp_id * M][0],
                                       COLS_PER_IT);
                for (int t = 0; t < BLOCK_ROW_WARPS; t++) {
                    wmma::load_matrix_sync(b_frag, 
                                           &shmem_b[t * N][0],
                                           COLS_PER_IT);
                    wmma::mma_sync(acc_frag[t], a_frag, b_frag, acc_frag[t]);
                }
                // if (num_new > 0 && j == 0 && num_old > 0 && global_base_by == 0) {
                //     int flag = atomicCAS(&for_check, 0, 1);
                //     if (!flag) {
                //         printf("check %d %d\n", num_new, num_old);
                //         auto *a_ptr = &shmem_a[warp_idy * M][warp_idx * N];
                //         for (int i = 0; i < 16; i++) {
                //             for (int j = 0; j < 16; j++) {
                //                 printf("%f ", (float)a_ptr[i * COLS_PER_IT + j]);
                //             } printf("\n");
                //         } printf("\n\n");
                //         for (int i = 0; i < a_frag.num_elements; i++) {
                //             printf("%f ", (float)a_frag.x[i]);
                //         } printf("\n\n");
                //         auto *b_ptr = &shmem_b[warp_idy * M][warp_idx * N];
                //         for (int i = 0; i < 16; i++) {
                //             for (int j = 0; j < 16; j++) {
                //                 printf("%f ", (float)b_ptr[i * COLS_PER_IT + j]);
                //             } printf("\n");
                //         } printf("\n\n");
                //         for (int i = 0; i < b_frag.num_elements; i++) {
                //             printf("%f ", (float)b_frag.x[i]);
                //         } printf("\n\n");
                //         for (int i = 0; i < acc_frag.num_elements; i++) {
                //             printf("%f ", (float)acc_frag.x[i]);
                //         } printf("\n\n");
                //     }
                // }
                // __syncthreads();
            }
            // __syncthreads();
            for (int k = 0; k < BLOCK_ROW_WARPS; k++) {
                for (int t = 0; t < acc_frag[k].num_elements; t++) {
                    acc_frag[k].x[t] *= -2.0f;
                }
            }
            for (int k = 0; k < BLOCK_ROW_WARPS; k++) {
                wmma::store_matrix_sync(&distances_cache[warp_id * BLOCK_ROW_WARPS * WMMA_M * WMMA_N + k * WMMA_N], 
                                        acc_frag[k], BLOCK_ROW_WARPS * WMMA_M, wmma::mem_row_major);
            }
            if (lane_id < 16) {
                squa_suma_cache[warp_id * 16 + lane_id] = squa_suma;
                squa_sumb_cache[warp_id * 16 + lane_id] = squa_sumb;
            }
            __syncthreads();
            // if (warp_id == 0 && global_base_by == 0) {
            //     int flag = atomicCAS(&for_check, 0, 1);
            //     if (!flag) {
            //         for (int i = 0; i < 64; i++) {
            //             printf("%f ", squa_suma_cache[i]);
            //         } printf("\n\n");
            //         for (int i = 0; i < 64; i++) {
            //             printf("%f ", squa_sumb_cache[i]);
            //         } printf("\n\n");
            //         int x = new_neighbors[0];
            //         float suma = 0;
            //         for (int i = 0; i < VEC_DIM; i++) {
            //             float val = vectors[x * VEC_DIM + i];
            //             suma += val * val;
            //         }
            //         printf("check %f\n", suma);
            //     }
            // }
            // back conflict!!!
            if (lane_id < 16) {
                int local_base_dcy = warp_id * WMMA_M;
                for (int k = 0; k < BLOCK_ROW_WARPS * WMMA_N; k++) {
                    int local_x = k;
                    int local_dcy = local_base_dcy + lane_id;
                    int global_dy = i * BLOCK_ROW_WARPS * WMMA_M + local_dcy;
                    int global_dx = j * BLOCK_ROW_WARPS * WMMA_N + local_x;
                    // if (warp_id >= 2) {
                    //     printf("check %d %d\n", global_dy, global_dx);
                    //     assert(warp_id < 2);
                    // }
                    if (global_dy < num_new && global_dx < num_old) {
                        // assert(warp_id < 2);
                        distances[global_dy * num_old + global_dx] = 
                            distances_cache[local_dcy * BLOCK_ROW_WARPS * WMMA_M + local_x] + 
                            squa_suma_cache[local_dcy] + squa_sumb_cache[local_x];
                    }
                }
            }
            // if (tx == 0) {
            //     for (int i = 0; i < 64; i++) {
            //         for (int j = 0; j < 64; j++) {
            //             int x = i, y = j;
            //             if (x < num_new && y < num_old) {
            //                 distances[x * num_old + y] = 
            //                     distances_cache[x * 64 + y];
            //             }
            //         }
            //     }
            // }
            __syncthreads();
            // __syncthreads();
            // if (global_base_ay < 16 && global_base_by < 16) {
            //     int flag = atomicCAS(&for_check, 0, 1);
            //     if (!flag) {
            //         printf("ffff %d %d %d\n", warp_id, global_base_ay, global_base_by);
            //         for (int i = 0; i < acc_frag[0].num_elements; i++) {
            //             printf("%f ", acc_frag[0].x[i]);
            //         } printf("\n\n");
            //         for (int i = 0; i < min((int)16, num_new); i++) {
            //             printf("%d ", new_neighbors[i]);
            //         } printf("\n\n");
            //         for (int i = 0; i < num_old; i++) {
            //             printf("%d ", old_neighbors[i]);
            //         } printf("\n\n");
            //         for (int i = 0; i < num_new; i++) {
            //             int x = new_neighbors[i];
            //             int y = old_neighbors[0];
            //             float suma = 0;
            //             half sumb = (half)0.0;
            //             for (int i = 0; i < VEC_DIM; i++) {
            //                 float a = vectors[x * VEC_DIM + i];
            //                 float b = vectors[y * VEC_DIM + i];
            //                 suma += a * b;
            //                 half aa = (half)a; half bb = (half)b;
            //                 sumb += aa * bb;
            //             }
            //             printf("Sum %d %f %f\n", i, suma, (float)sumb);
            //         }
            //         for (int i = 0; i < 64; i++) {
            //             for (int j = 0; j < 64; j++) {
            //                 printf("%f ", distances_cache[i * 64 + j]);
            //             } printf("\n");
            //         } printf("\n\n");
            //     }
            // }
            // __syncthreads();
            // if (lane_id == 0) {
            //     float *start = &distances_cache[(warp_id / 3) * 48 * 16 + (warp_id % 3) * 16];
            //     for (int i = 0; i < 16; i++) {
            //         for (int j = 0; j < 16; j++) {
            //             if (global_base_ay + i < num_new && global_base_by + j < num_old) {
            //                 distances[(global_base_ay + i) * num_old + global_base_by + j] = start[i * 48 + j];
            //             }
            //         }
            //     }
            // }
            // global_base_ay * num_old + global_base_by
        }
    }
}

__global__ void NewOldNeighborsCompareKernel(ResultElement *knn_graph, int *global_locks, 
                                             const float *vectors,
                                             const int *edges_new, const int *dest_new,
                                             const int num_new_max, 
                                             const int *edges_old, const int *dest_old,
                                             const int num_old_max) {
    extern __shared__ char buffer[];

    __shared__ float *distances;
    __shared__ int *neighbors, *local_locks;
    __shared__ ResultElement *knn_graph_cache;

    __shared__ int pos_gnew, pos_gold, num_new, num_old;

    __shared__ half shmem_a[BLOCK_ROW_WARPS * WMMA_M][BLOCK_COL_WARPS * WMMA_K];
    __shared__ half shmem_b[BLOCK_ROW_WARPS * WMMA_N][BLOCK_COL_WARPS * WMMA_K];

    int neighb_num_max = num_new_max + num_old_max;
    int tx = threadIdx.x;
    if (tx == 0) {
        distances = (float *)buffer;
        neighbors = 
            (int *)((char *)buffer + (num_new_max * num_old_max) * sizeof(float));
        local_locks = 
            (int *)((char *)neighbors + neighb_num_max * sizeof(int));
        knn_graph_cache =
            (ResultElement *)((char *)local_locks + neighb_num_max * sizeof(int));
    }
    __syncthreads();

    int list_id = blockIdx.x;
    int block_dim_x = blockDim.x;

    if (tx < neighb_num_max) {
        local_locks[tx] = 0;
    }

    if (tx == 0) {
        int next_pos = edges_new[list_id + 1];
        int now_pos = edges_new[list_id];
        num_new = next_pos - now_pos;
        pos_gnew = now_pos;
    } else if (tx == 32) {
        int next_pos = edges_old[list_id + 1];
        int now_pos = edges_old[list_id];
        num_old = next_pos - now_pos;
        pos_gold = now_pos;
    }
    __syncthreads();
    int neighb_num = num_new + num_old;
    if (tx < num_new) {
        neighbors[tx] = dest_new[pos_gnew + tx];
    } else if (tx >= num_new && tx < neighb_num) {
        neighbors[tx] = dest_old[pos_gold + tx - num_new];
    }
    __syncthreads();

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("new-old calc. num. %d %d %d\n", num_new, num_old, neighb_num);
    } __syncthreads();

    GetDistancesWMMA(distances, vectors, 
                     neighbors, num_new, neighbors + num_new, num_old,
                     shmem_a, shmem_b);
    __syncthreads();
    // if (num_old >= 2) {
    //     int flag = atomicCAS(&for_check, 0, 1);
    //     if (!flag) {
    //         printf("\nCheck distance %d %d: \n", num_new, num_old);
    //         for (int i = 0; i < num_new; i++) {
    //             for (int j = 0; j < num_old; j++) {
    //                 float distance = 0;
    //                 int x = *(neighbors + i);
    //                 int y = *(neighbors + j + num_new);
    //                 for (int k = 0; k < VEC_DIM; k++) {
    //                     float diff = vectors[x * VEC_DIM + k] - 
    //                                  vectors[y * VEC_DIM + k];
    //                     distance += diff * diff;
    //                 }
    //                 printf("%f ", distance);
    //             } 
    //         } printf("\n\n");
    //         for (int i = 0; i < num_new * num_old; i++) {
    //             printf("%f ", distances[i]);
    //         } printf("\n\n");
    //     }
    // }
    // return;
    int calc_num = num_new * num_old;
    // int num_it = GetItNum(NEIGHB_NUM_PER_LIST, NEIGHB_CACHE_NUM);
    const int num_it = 1;
    for (int i = 0; i < num_it; i++) {
        // Read list to cache
        int num_it2 = GetItNum(neighb_num * NEIGHB_CACHE_NUM, block_dim_x);
        for (int j = 0; j < num_it2; j++) {
            int pos = j * block_dim_x + tx;
            if (pos < neighb_num * NEIGHB_CACHE_NUM)
                knn_graph_cache[pos] = ResultElement(1e10, 33333333);
        }
        int list_size = NEIGHB_CACHE_NUM;
        int num_it3 = GetItNum(neighb_num, block_dim_x / THREADS_PER_LIST);
        for (int j = 0; j < num_it3; j++) {
            int list_id = j * (block_dim_x / THREADS_PER_LIST) + 
                          tx / THREADS_PER_LIST;
            if (list_id >= neighb_num) continue;
            if (list_id < num_new) {
                UpdateLocalNewKNNLists(knn_graph_cache, list_id, list_size, 
                                       neighbors + num_new, num_old, 
                                       distances, calc_num);
            } else {
                UpdateLocalOldKNNLists(knn_graph_cache, list_id, list_size, 
                                       neighbors, num_new, 
                                       neighbors + num_new, num_old, 
                                       distances, calc_num, vectors);
            }
       }
        __syncthreads();
        MergeLocalGraphWithGlobalGraph(knn_graph_cache, list_size, neighbors,
                                       neighb_num, knn_graph, global_locks);
        __syncthreads();
    }
}

pair<int*, int*> ReadGraphToGlobalMemory(const Graph& graph) {
    int pos = 0;
    vector<int> edges, dest;
    for (int i = 0; i < graph.size(); i++) {
        edges.push_back(pos);
        // dest.push_back(graph[i].size());
        // pos++;
        for (int j = 0; j < graph[i].size(); j++) {
            dest.push_back(graph[i][j]);
            pos++;
        }
    }
    edges.push_back(pos);

    int *edges_dev, *dest_dev;
    hipError_t cuda_status0, cuda_status1;
    cuda_status0 = hipMalloc(&edges_dev, edges.size() * sizeof(int));
    cuda_status1 = hipMalloc(&dest_dev, dest.size() * sizeof(int));
    if (cuda_status0 != hipSuccess || cuda_status1 != hipSuccess) {
        cerr << "CudaMalloc failed" << endl;
        exit(-1);
    }

    cuda_status0 = hipMemcpy(edges_dev, edges.data(), 
                              edges.size() * sizeof(int), hipMemcpyHostToDevice);
    cuda_status1 = hipMemcpy(dest_dev, dest.data(), 
                              dest.size() * sizeof(int), hipMemcpyHostToDevice);
    if (cuda_status0 != hipSuccess || cuda_status1 != hipSuccess) {
        cerr << "CudaMemcpy failed" << endl;
        exit(-1);
    }
    return make_pair(edges_dev, dest_dev);
}

__global__ void TestKernel(ResultElement* knn_graph) {
    for (int i = 0; i < 10000 * 30; i++) {
        if (knn_graph[i].distance == 0 && knn_graph[i].label == 0) {
            printf("check %d %f\n", i, knn_graph[i].distance);
        }
    }
    return;
}

ResultElement* ReadKNNGraphToGlobalMemory(const vector<vector<gpuknn::NNDItem>> &knn_graph) {
    int k = knn_graph[0].size();
    ResultElement *knn_graph_dev;
    ResultElement *knn_graph_host = new ResultElement[knn_graph.size() * k];
    int idx = 0;
    for (int i = 0; i < knn_graph.size(); i++) {
        for (int j = 0; j < k; j++) {
            const auto &item = knn_graph[i][j];
            knn_graph_host[idx++] = ResultElement(item.distance, item.id);
        }
    }

    auto cuda_status = hipMalloc(&knn_graph_dev, 
                                  knn_graph.size() * k * sizeof(ResultElement));
    if (cuda_status != hipSuccess) {
        cerr << "knn_graph hipMalloc failed." << endl;
        exit(-1);
    }
    cuda_status = hipMemcpy(knn_graph_dev, knn_graph_host, 
                             knn_graph.size() * k * sizeof(ResultElement), 
                             hipMemcpyHostToDevice);
    if (cuda_status != hipSuccess) {
        cerr << hipGetErrorString(cuda_status) << endl;
        cerr << "knn_graph hipMemcpyHostToDevice failed." << endl;
        exit(-1);
    }
    delete [] knn_graph_host;
    return knn_graph_dev;
}

void ToHostGraph(vector<vector<gpuknn::NNDItem>> *origin_knn_graph_ptr,
                 const ResultElement *knn_graph, const int size, const int neighb_num) {
    auto &origin_knn_graph = *origin_knn_graph_ptr;
    vector<gpuknn::NNDItem> neighb_list;
    for (int i = 0; i < size; i++) {
        neighb_list.clear();
        for (int j = 0; j < neighb_num; j++) {
            ResultElement tmp = knn_graph[i * neighb_num + j];
            neighb_list.emplace_back(tmp.label, false, tmp.distance);
        }
        for (int j = 0; j < neighb_num; j++) {
            for (int k = 0; k < neighb_num; k++) {
                if (neighb_list[j].id == origin_knn_graph[i][k].id) {
                    neighb_list[j].visited = true;
                    break;
                }
            }
        }
        origin_knn_graph[i] = neighb_list;
    }
}

int GetMaxListSize(const Graph &g) {
    int res = 0;
    for (const auto &list : g) {
        res = max((int)list.size(), res);
    }
    return res;
}

void UpdateGraph(vector<vector<gpuknn::NNDItem>> *origin_knn_graph_ptr, 
                 float* vectors_dev, 
                 const Graph& newg, const Graph& oldg, const int k) {
    auto &origin_knn_graph = *origin_knn_graph_ptr;

    int *edges_dev_new, *dest_dev_new;
    tie(edges_dev_new, dest_dev_new) = ReadGraphToGlobalMemory(newg);

    int *edges_dev_old, *dest_dev_old;
    tie(edges_dev_old, dest_dev_old) = ReadGraphToGlobalMemory(oldg);

    size_t g_size = newg.size();

    hipError_t cuda_status;
    ResultElement *knn_graph_dev, *knn_graph = new ResultElement[g_size * k];
    knn_graph_dev = ReadKNNGraphToGlobalMemory(origin_knn_graph);

    int *global_locks_dev;
    hipMalloc(&global_locks_dev, g_size * sizeof(int));
    vector<int> zeros(g_size);
    hipMemcpy(global_locks_dev, zeros.data(), g_size * sizeof(int),
               hipMemcpyHostToDevice);
    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess) {
        cerr << hipGetErrorString(cuda_status) << endl;
        cerr << "Initiate failed" << endl;
        exit(-1);
    }

    dim3 block_size(512);
    dim3 grid_size(g_size);
    // cerr << "Start kernel." << endl;
    const int num_new_max = GetMaxListSize(newg);
    const int num_old_max = GetMaxListSize(oldg);
    size_t shared_memory_size = 
        num_new_max * VEC_DIM * sizeof(float) + 
        (num_new_max * (num_new_max - 1) / 2) * sizeof(float) +
        num_new_max * 2 * sizeof(int) + 
        num_new_max * NEIGHB_CACHE_NUM * sizeof(ResultElement);

    NewNeighborsCompareKernel<<<grid_size, block_size, shared_memory_size>>>
        (knn_graph_dev, global_locks_dev, vectors_dev,
         edges_dev_new, dest_dev_new, num_new_max);
    hipDeviceSynchronize();

    block_size = dim3(128);
    int neighb_num_max = num_new_max + num_old_max;
    shared_memory_size = (num_new_max * num_old_max) * sizeof(float) + 
                         neighb_num_max * 2 * sizeof(int) + 
                         neighb_num_max * NEIGHB_CACHE_NUM * sizeof(ResultElement);
    cerr << "shmem kernel2 size: " << shared_memory_size << endl;
    auto start = chrono::steady_clock::now();
    NewOldNeighborsCompareKernel<<<grid_size, block_size, shared_memory_size>>>
        (knn_graph_dev, global_locks_dev, vectors_dev, 
         edges_dev_new, dest_dev_new, num_new_max,
         edges_dev_old, dest_dev_old, num_old_max);

    // LocalDistCompareKernel<<<grid_size, block_size>>>(knn_graph_dev, 
    //                                                   global_locks_dev,
    //                                                   vectors_dev,
    //                                                   edges_dev_new, dest_dev_new, 
    //                                                   edges_dev_old, dest_dev_old);
    hipDeviceSynchronize();
    auto end = chrono::steady_clock::now();
    cerr << "Kernel 2 costs: "
         << (float)chrono::duration_cast<chrono::microseconds>(end - start).count() / 1e6
         << endl;

    cuda_status = hipGetLastError();

    if (cuda_status != hipSuccess) {
        cerr << hipGetErrorString(cuda_status) << endl;
        cerr << "Kernel failed" << endl;
        exit(-1);
    }
    // cerr << "End kernel." << endl;
    cuda_status = hipMemcpy(knn_graph, knn_graph_dev, 
                             g_size * k * sizeof(ResultElement), hipMemcpyDeviceToHost);
    if (cuda_status != hipSuccess) {
        cerr << hipGetErrorString(cuda_status) << endl;
        cerr << "knn_graph hipMemcpy failed" << endl;
        exit(-1);
    }

    ToHostGraph(&origin_knn_graph, knn_graph, g_size, k);

    delete [] knn_graph;
    hipFree(edges_dev_new);
    hipFree(dest_dev_new);
    hipFree(edges_dev_old);
    hipFree(dest_dev_old);
    hipFree(knn_graph_dev);
}

namespace gpuknn {
    vector<vector<NNDItem>> NNDescent(const float* vectors, const int vecs_size, const int vecs_dim) {
        int k = NEIGHB_NUM_PER_LIST;
        int iteration = 6;
        auto cuda_status = hipSetDevice(DEVICE_ID);

        float* vectors_dev;
        hipMalloc(&vectors_dev, (size_t)vecs_size * vecs_dim * sizeof(float));
        hipMemcpy(vectors_dev, vectors, 
                   (size_t)vecs_size * vecs_dim * sizeof(float),
                   hipMemcpyHostToDevice);

        if (cuda_status != hipSuccess) {
            cerr << hipGetErrorString(cuda_status) << endl;
            cerr << "hipSetDevice failed" << endl;
            exit(-1);
        }
        Graph result(vecs_size);
        vector<vector<NNDItem>> g(vecs_size);
        vector<int> tmp_vec;

        for (int i = 0; i < vecs_size; i++) {
            vector<int> exclusion = {i};
            xmuknn::GenerateRandomSequence(tmp_vec, k, vecs_size, exclusion);
            for (int j = 0; j < k; j++) {
                int nb_id = tmp_vec[j];
                g[i].emplace_back(nb_id, false, 1e10);
            }
        }

        #pragma omp parallel for
        for (int i = 0; i < vecs_size; i++) {
            for (int j = 0; j < k; j++) {
                g[i][j].distance = 
                    GetDistance(vectors + (size_t)i * vecs_dim, 
                                vectors + (size_t)g[i][j].id * vecs_dim,
                                vecs_dim);
            }
        }

        #pragma omp parallel for
        for (int i = 0; i < g.size(); i++) {
            sort(g[i].begin(), g[i].end(), [](NNDItem a, NNDItem b) {
                    if (fabs(a.distance - b.distance) < 1e-10) return a.id < b.id;
                    return a.distance < b.distance;
                 });
        }

        float kernel_costs = 0;
        Graph newg, oldg;
        float get_nb_graph_time = 0;
        auto sum_start = chrono::steady_clock::now();
        for (int t = 0; t < iteration; t++) {
            cerr << "Start generating NBGraph." << endl;
            auto start = chrono::steady_clock::now();
            GetNBGraph(&newg, &oldg, g, vectors, vecs_size, vecs_dim);
            auto end = chrono::steady_clock::now();
            float tmp_time = 
                (float)chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6;
            get_nb_graph_time += tmp_time;
            cerr << "GetNBGraph costs "
                 << tmp_time
                 << endl;

            start = chrono::steady_clock::now();
            vector<pair<float, int>> tmp_result;
            // long long update_times = 0;
            UpdateGraph(&g, vectors_dev, newg, oldg, k);
            end = chrono::steady_clock::now();
            float kernel_tmp_costs = (float)chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e6;
            kernel_costs += kernel_tmp_costs;
            cerr << "Kernel costs "
                 << kernel_tmp_costs
                 << endl;
            cerr << endl;
        }
        auto sum_end = chrono::steady_clock::now();
        float sum_costs = (float)chrono::duration_cast<std::chrono::microseconds>(sum_end - sum_start).count() / 1e6;
        // sift10k in cpu should be 0.6s;
        cerr << "All kernel costs: " << kernel_costs << endl;
        cerr << "Get NB graph costs: " << get_nb_graph_time << endl; 
        cerr << "All procedure costs: " << sum_costs << endl;
        return g;
    }
}

#endif