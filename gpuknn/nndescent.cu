#include "hip/hip_runtime.h"
#include <assert.h>
#include <hiprand.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <algorithm>
#include <bitset>
#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <mutex>
#include <tuple>
#include <utility>
#include <vector>

#include "../tools/distfunc.hpp"
#include "../tools/nndescent_element.cuh"
#include "../xmuknn.h"
#include "hip/hip_runtime.h"
#include ""
#include "knncuda_tools.cuh"
#include "nndescent.cuh"

#ifdef __INTELLISENSE__
#include "../intellisense_cuda_intrinsics.h"
#endif

using namespace std;
using namespace xmuknn;
#define DEVICE_ID 0
#define FULL_MASK 0xffffffff
#define VERBOSE 0
// #define DONT_TILE 0
// #define INSERT_MIN_ONLY 1

__device__ int for_check = 0;

__global__ void PrepareGraph(int *graph_new_dev, int *newg_list_size_dev,
                             int *graph_old_dev, int *oldg_list_size_dev,
                             NNDElement *knn_graph, int graph_size) {
  __shared__ int new_elements_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int cache1_size;
  __shared__ int old_elements_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int cache2_size;
  int list_id = blockIdx.x;
  int knng_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int nn_list_base_pos = list_id * (SAMPLE_NUM * 2);
  int tx = threadIdx.x;
  if (tx == 0) {
    cache1_size = cache2_size = 0;
  }
  __syncthreads();
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int nth = i * WARP_SIZE + tx;
    int local_pos =
        nth % NEIGHB_BLOCKS_NUM * WARP_SIZE + nth / NEIGHB_BLOCKS_NUM;
    if (local_pos < NEIGHB_NUM_PER_LIST) {
      NNDElement elem = knn_graph[knng_base_pos + local_pos];
      if (elem.IsNew()) {
        int pos = atomicAdd(&cache1_size, 1);
        new_elements_cache[pos] = elem.label();
      } else {
        int pos = atomicAdd(&cache2_size, 1);
        old_elements_cache[pos] = elem.label();
      }
    }
  }
  __syncthreads();
  if (tx == 0) {
    cache1_size = min(cache1_size, SAMPLE_NUM);
    cache2_size = min(cache2_size, SAMPLE_NUM);
  }
  __syncthreads();
  if (tx == 0) {
    newg_list_size_dev[list_id] = cache1_size;
    oldg_list_size_dev[list_id] = cache2_size;
  }
  it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      graph_new_dev[nn_list_base_pos + local_pos] =
          new_elements_cache[local_pos];
    }
    if (local_pos < cache2_size) {
      graph_old_dev[nn_list_base_pos + local_pos] =
          old_elements_cache[local_pos];
    }
  }
  __syncthreads();
}

__global__ void PrepareGraphForMerge(int *graph_new_dev,
                                     int *newg_list_size_dev,
                                     int *graph_old_dev,
                                     int *oldg_list_size_dev,
                                     NNDElement *knn_graph, int split_pos,
                                     int graph_size) {
  __shared__ int new_elements_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int cache1_size;
  __shared__ int old_elements_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int cache2_size;
  int list_id = blockIdx.x;
  int knng_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int nn_list_base_pos = list_id * (SAMPLE_NUM * 2);
  int tx = threadIdx.x;
  if (tx == 0) {
    cache1_size = cache2_size = 0;
  }
  __syncthreads();
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int nth = i * WARP_SIZE + tx;
    int local_pos =
        nth % NEIGHB_BLOCKS_NUM * WARP_SIZE + nth / NEIGHB_BLOCKS_NUM;
    if (local_pos < NEIGHB_NUM_PER_LIST) {
      NNDElement elem = knn_graph[knng_base_pos + local_pos];
      if (elem.IsNew()) {
        int pos = atomicAdd(&cache1_size, 1);
        new_elements_cache[pos] = elem.label();
      } else {
        if (list_id < split_pos) {
          if (elem.label() >= split_pos) {
            continue;
          }
        } else if (list_id >= split_pos) {
          if (elem.label() < split_pos) {
            continue;
          }
        }
        int pos = atomicAdd(&cache2_size, 1);
        old_elements_cache[pos] = elem.label();
      }
    }
  }
  __syncthreads();
  if (tx == 0) {
    cache1_size = min(cache1_size, SAMPLE_NUM);
    cache2_size = min(cache2_size, SAMPLE_NUM);
  }
  __syncthreads();
  if (tx == 0) {
    newg_list_size_dev[list_id] = cache1_size;
    oldg_list_size_dev[list_id] = cache2_size;
  }
  it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      graph_new_dev[nn_list_base_pos + local_pos] =
          new_elements_cache[local_pos];
    }
    if (local_pos < cache2_size) {
      graph_old_dev[nn_list_base_pos + local_pos] =
          old_elements_cache[local_pos];
    }
  }
  __syncthreads();
}

template <typename T>
__device__ void Swap(T &a, T &b) {
  T c = a;
  a = b;
  b = c;
}

template <typename T>
__device__ void InsertSort(T *a, const int length) {
  for (int i = 1; i < length; i++) {
    for (int j = i - 1; j >= 0 && a[j + 1] < a[j]; j--) {
      Swap(a[j], a[j + 1]);
    }
  }
}

__device__ __forceinline__ NNDElement XorSwap(NNDElement x, int mask, int dir) {
  NNDElement y;
  y.distance_ = __shfl_xor_sync(FULL_MASK, x.distance_, mask, WARP_SIZE);
  y.label_ = __shfl_xor_sync(FULL_MASK, x.label_, mask, WARP_SIZE);
  return x < y == dir ? y : x;
}

__device__ __forceinline__ int XorSwap(int x, int mask, int dir) {
  int y;
  y = __shfl_xor_sync(FULL_MASK, x, mask, WARP_SIZE);
  return x < y == dir ? y : x;
}

__device__ __forceinline__ uint Bfe(uint lane_id, uint pos) {
  uint res;
  asm("bfe.u32 %0,%1,%2,%3;" : "=r"(res) : "r"(lane_id), "r"(pos), "r"(1));
  return res;
}

template <typename T>
__device__ __forceinline__ void BitonicSort(T *sort_element_ptr,
                                            const int lane_id) {
  auto &sort_elem = *sort_element_ptr;
  sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 1) ^ Bfe(lane_id, 0));
  sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 2) ^ Bfe(lane_id, 1));
  sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 2) ^ Bfe(lane_id, 0));
  sort_elem = XorSwap(sort_elem, 0x04, Bfe(lane_id, 3) ^ Bfe(lane_id, 2));
  sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 3) ^ Bfe(lane_id, 1));
  sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 3) ^ Bfe(lane_id, 0));
  sort_elem = XorSwap(sort_elem, 0x08, Bfe(lane_id, 4) ^ Bfe(lane_id, 3));
  sort_elem = XorSwap(sort_elem, 0x04, Bfe(lane_id, 4) ^ Bfe(lane_id, 2));
  sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 4) ^ Bfe(lane_id, 1));
  sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 4) ^ Bfe(lane_id, 0));
  sort_elem = XorSwap(sort_elem, 0x10, Bfe(lane_id, 4));
  sort_elem = XorSwap(sort_elem, 0x08, Bfe(lane_id, 3));
  sort_elem = XorSwap(sort_elem, 0x04, Bfe(lane_id, 2));
  sort_elem = XorSwap(sort_elem, 0x02, Bfe(lane_id, 1));
  sort_elem = XorSwap(sort_elem, 0x01, Bfe(lane_id, 0));
  return;
}

template <typename T>
__device__ int MergeList(T *A, const int m, T *B, const int n, T *C) {
  int i = 0, j = 0, cnt = 0;
  while ((i < m) && (j < n)) {
    if (A[i] <= B[j]) {
      C[cnt++] = A[i++];
      if (cnt >= NEIGHB_NUM_PER_LIST) goto EXIT;
    } else {
      C[cnt++] = B[j++];
      if (cnt >= NEIGHB_NUM_PER_LIST) goto EXIT;
    }
  }

  if (i == m) {
    for (; j < n; j++) {
      C[cnt++] = B[j];
      if (cnt >= NEIGHB_NUM_PER_LIST) goto EXIT;
    }
  } else {
    for (; i < m; i++) {
      C[cnt++] = A[i];
      if (cnt >= NEIGHB_NUM_PER_LIST) goto EXIT;
    }
  }
EXIT:
  return cnt;
}

__device__ int RemoveDuplicates(int *nums, int nums_size) {
  if (nums_size < 2) return nums_size;
  int a = 0, b = 1;
  while (b < nums_size)
    if (nums[b++] > nums[a]) nums[++a] = nums[b - 1];
  return (a + 1);
}

__global__ void PrepareReverseGraph(int *graph_new_dev, int *newg_list_size_dev,
                                    int *newg_revlist_size_dev,
                                    int *graph_old_dev, int *oldg_list_size_dev,
                                    int *oldg_revlist_size_dev) {
  __shared__ int new_elements_cache[SAMPLE_NUM];
  __shared__ int cache1_size;
  __shared__ int old_elements_cache[SAMPLE_NUM];
  __shared__ int cache2_size;
  int tx = threadIdx.x;
  int list_id = blockIdx.x;
  int knng_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int nn_list_base_pos = list_id * (SAMPLE_NUM * 2);
  if (tx == 0) {
    cache1_size = newg_list_size_dev[list_id];
    cache2_size = oldg_list_size_dev[list_id];
  }
  __syncthreads();
  int it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      new_elements_cache[local_pos] =
          graph_new_dev[nn_list_base_pos + local_pos];
    }
    if (local_pos < cache2_size) {
      old_elements_cache[local_pos] =
          graph_old_dev[nn_list_base_pos + local_pos];
    }
  }
  __syncthreads();
  it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      int rev_list_id = new_elements_cache[local_pos];
      int pos = SAMPLE_NUM;
      pos += atomicAdd(&newg_revlist_size_dev[rev_list_id], 1);
      // printf("%d %d %d\n", pos, rev_list_id, list_id);
      if (pos >= SAMPLE_NUM * 2)
        atomicExch(&newg_revlist_size_dev[rev_list_id], SAMPLE_NUM);
      else
        graph_new_dev[rev_list_id * (SAMPLE_NUM * 2) + pos] = list_id;
    }
    if (local_pos < cache2_size) {
      int rev_list_id = old_elements_cache[local_pos];
      int pos = SAMPLE_NUM;
      pos += atomicAdd(&oldg_revlist_size_dev[rev_list_id], 1);
      if (pos >= SAMPLE_NUM * 2)
        atomicExch(&oldg_revlist_size_dev[rev_list_id], SAMPLE_NUM);
      else
        graph_old_dev[rev_list_id * (SAMPLE_NUM * 2) + pos] = list_id;
    }
  }
}

__global__ void PrepareReverseGraphForMerge(
    int *graph_new_dev, int *newg_list_size_dev, int *newg_revlist_size_dev,
    int *graph_old_dev, int *oldg_list_size_dev, int *oldg_revlist_size_dev,
    const int split_pos) {
  __shared__ int new_elements_cache[SAMPLE_NUM];
  __shared__ int cache1_size;
  __shared__ int old_elements_cache[SAMPLE_NUM];
  __shared__ int cache2_size;
  int tx = threadIdx.x;
  int list_id = blockIdx.x;
  int knng_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int nn_list_base_pos = list_id * (SAMPLE_NUM * 2);
  if (tx == 0) {
    cache1_size = newg_list_size_dev[list_id];
    cache2_size = oldg_list_size_dev[list_id];
  }
  __syncthreads();
  int it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      new_elements_cache[local_pos] =
          graph_new_dev[nn_list_base_pos + local_pos];
    }
    if (local_pos < cache2_size) {
      old_elements_cache[local_pos] =
          graph_old_dev[nn_list_base_pos + local_pos];
    }
  }
  __syncthreads();
  it_num = GetItNum(SAMPLE_NUM, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < cache1_size) {
      int rev_list_id = new_elements_cache[local_pos];
      if (list_id < split_pos && rev_list_id >= split_pos) continue;
      int pos = SAMPLE_NUM;
      pos += atomicAdd(&newg_revlist_size_dev[rev_list_id], 1);
      // printf("%d %d %d\n", pos, rev_list_id, list_id);
      if (pos >= SAMPLE_NUM * 2)
        atomicExch(&newg_revlist_size_dev[rev_list_id], SAMPLE_NUM);
      else
        graph_new_dev[rev_list_id * (SAMPLE_NUM * 2) + pos] = list_id;
    }
    if (local_pos < cache2_size) {
      int rev_list_id = old_elements_cache[local_pos];
      if (list_id >= split_pos && rev_list_id < split_pos) continue;
      int pos = SAMPLE_NUM;
      pos += atomicAdd(&oldg_revlist_size_dev[rev_list_id], 1);
      if (pos >= SAMPLE_NUM * 2)
        atomicExch(&oldg_revlist_size_dev[rev_list_id], SAMPLE_NUM);
      else
        graph_old_dev[rev_list_id * (SAMPLE_NUM * 2) + pos] = list_id;
    }
  }
}

__global__ void ShrinkGraph(int *graph_new_dev, int *newg_list_size_dev,
                            int *newg_revlist_size_dev, int *graph_old_dev,
                            int *oldg_list_size_dev,
                            int *oldg_revlist_size_dev) {
  __shared__ int new_elements_cache[SAMPLE_NUM * 2];
  __shared__ int newg_list_size, newg_revlist_size;
  __shared__ int old_elements_cache[SAMPLE_NUM * 2];
  __shared__ int sorted_elements_cache[32];
  __shared__ int merged_list_cache[NEIGHB_NUM_PER_LIST];
  __shared__ int oldg_list_size, oldg_revlist_size;
  int tx = threadIdx.x;
  int list_id = blockIdx.x;
  int nn_list_base_pos = list_id * (SAMPLE_NUM * 2);
  int lane_id = tx % WARP_SIZE;

  if (tx == 0) {
    newg_list_size = newg_list_size_dev[list_id];
    oldg_list_size = oldg_list_size_dev[list_id];
    newg_revlist_size = newg_revlist_size_dev[list_id];
    oldg_revlist_size = oldg_revlist_size_dev[list_id];
  }
  __syncthreads();
  int it_num = GetItNum(SAMPLE_NUM * 2, WARP_SIZE);
  int list_new_size = 0, list_old_size = 0;
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;

    int sort_elem = LARGE_INT;
    if ((local_pos < newg_list_size) ||
        (local_pos >= SAMPLE_NUM &&
         local_pos < SAMPLE_NUM + newg_revlist_size)) {
      sort_elem = graph_new_dev[nn_list_base_pos + local_pos];
    }
    BitonicSort(&sort_elem, lane_id);
    sorted_elements_cache[lane_id] = sort_elem;
    if (lane_id == 0) {
      list_new_size =
          MergeList(new_elements_cache, list_new_size, sorted_elements_cache,
                    WARP_SIZE, merged_list_cache);
    }
    list_new_size = __shfl_sync(FULL_MASK, list_new_size, 0);
    int copy_it_num = GetItNum(list_new_size, WARP_SIZE);
    for (int j = 0; j < copy_it_num; j++) {
      int pos = j * WARP_SIZE + lane_id;
      if (pos >= SAMPLE_NUM * 2) break;
      new_elements_cache[pos] = merged_list_cache[pos];
    }

    sort_elem = LARGE_INT;
    if ((local_pos < oldg_list_size) ||
        (local_pos >= SAMPLE_NUM &&
         local_pos < SAMPLE_NUM + oldg_revlist_size)) {
      sort_elem = graph_old_dev[nn_list_base_pos + local_pos];
    }
    BitonicSort(&sort_elem, lane_id);
    sorted_elements_cache[lane_id] = sort_elem;
    if (lane_id == 0) {
      list_old_size =
          MergeList(old_elements_cache, list_old_size, sorted_elements_cache,
                    WARP_SIZE, merged_list_cache);
    }
    list_old_size = __shfl_sync(FULL_MASK, list_old_size, 0);
    copy_it_num = GetItNum(list_old_size, WARP_SIZE);
    for (int j = 0; j < copy_it_num; j++) {
      int pos = j * WARP_SIZE + lane_id;
      if (pos >= SAMPLE_NUM * 2) break;
      old_elements_cache[pos] = merged_list_cache[pos];
    }
  }
  __syncthreads();
  if (tx == 0) {
    newg_list_size = RemoveDuplicates(new_elements_cache, list_new_size);
    newg_list_size -= (new_elements_cache[newg_list_size - 1] == LARGE_INT);
    oldg_list_size = RemoveDuplicates(old_elements_cache, list_old_size);
    oldg_list_size -= (old_elements_cache[oldg_list_size - 1] == LARGE_INT);
    for (int i = 0; i < newg_list_size; i++) {
      for (int j = 0; j < oldg_list_size; j++) {
        if (new_elements_cache[i] == old_elements_cache[j]) {
          new_elements_cache[i] = -1;
          break;
        }
      }
    }
    int pos = 0;
    for (int i = 0; i < newg_list_size; i++) {
      if (new_elements_cache[i] != -1) {
        new_elements_cache[pos++] = new_elements_cache[i];
      }
    }
    newg_list_size = pos;
  }
  __syncthreads();
  it_num = GetItNum(SAMPLE_NUM * 2, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int local_pos = i * WARP_SIZE + tx;
    if (local_pos < newg_list_size) {
      graph_new_dev[nn_list_base_pos + local_pos] =
          new_elements_cache[local_pos];
    }
    if (local_pos < oldg_list_size) {
      graph_old_dev[nn_list_base_pos + local_pos] =
          old_elements_cache[local_pos];
    }
  }

  newg_list_size_dev[list_id] = newg_list_size;
  oldg_list_size_dev[list_id] = oldg_list_size;
}

void PrepareForUpdate(int *graph_new_dev, int *newg_list_size_dev,
                      int *newg_revlist_size_dev, int *graph_old_dev,
                      int *oldg_list_size_dev, int *oldg_revlist_size_dev,
                      NNDElement *knn_graph_dev, int graph_size) {
  auto start = chrono::steady_clock::now();
  hipMemset(newg_list_size_dev, 0, graph_size * sizeof(int));
  hipMemset(oldg_list_size_dev, 0, graph_size * sizeof(int));
  hipMemset(newg_revlist_size_dev, 0, graph_size * sizeof(int));
  hipMemset(oldg_revlist_size_dev, 0, graph_size * sizeof(int));
  dim3 grid_size(graph_size);
  dim3 block_size(32);
  PrepareGraph<<<grid_size, block_size>>>(graph_new_dev, newg_list_size_dev,
                                          graph_old_dev, oldg_list_size_dev,
                                          knn_graph_dev, graph_size);
  hipDeviceSynchronize();
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Prepare kernel failed." << endl;
    exit(-1);
  }
  PrepareReverseGraph<<<grid_size, block_size>>>(
      graph_new_dev, newg_list_size_dev, newg_revlist_size_dev, graph_old_dev,
      oldg_list_size_dev, oldg_revlist_size_dev);
  hipDeviceSynchronize();
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "PrepareReverseGraph kernel failed." << endl;
    exit(-1);
  }
  ShrinkGraph<<<grid_size, block_size>>>(
      graph_new_dev, newg_list_size_dev, newg_revlist_size_dev, graph_old_dev,
      oldg_list_size_dev, oldg_revlist_size_dev);
  hipDeviceSynchronize();
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "ShrinkGraph kernel failed." << endl;
    exit(-1);
  }
  auto end = chrono::steady_clock::now();
  if (VERBOSE) {
    cerr << "Prepare kernel costs: "
         << (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
                    .count() /
                1e6
         << endl;
  }
}

void PrepareForUpdateForMerge(int *graph_new_dev, int *newg_list_size_dev,
                              int *newg_revlist_size_dev, int *graph_old_dev,
                              int *oldg_list_size_dev,
                              int *oldg_revlist_size_dev,
                              NNDElement *knn_graph_dev, int split_pos,
                              int graph_size) {
  auto start = chrono::steady_clock::now();
  hipMemset(newg_list_size_dev, 0, graph_size * sizeof(int));
  hipMemset(oldg_list_size_dev, 0, graph_size * sizeof(int));
  hipMemset(newg_revlist_size_dev, 0, graph_size * sizeof(int));
  hipMemset(oldg_revlist_size_dev, 0, graph_size * sizeof(int));
  dim3 grid_size(graph_size);
  dim3 block_size(32);
  PrepareGraphForMerge<<<grid_size, block_size>>>(
      graph_new_dev, newg_list_size_dev, graph_old_dev, oldg_list_size_dev,
      knn_graph_dev, split_pos, graph_size);
  hipDeviceSynchronize();
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Prepare kernel failed." << endl;
    exit(-1);
  }
  // PrepareReverseGraphForMerge<<<grid_size, block_size>>>(
  //     graph_new_dev, newg_list_size_dev, newg_revlist_size_dev, graph_old_dev,
  //     oldg_list_size_dev, oldg_revlist_size_dev, split_pos);
  PrepareReverseGraph<<<grid_size, block_size>>>(
      graph_new_dev, newg_list_size_dev, newg_revlist_size_dev, graph_old_dev,
      oldg_list_size_dev, oldg_revlist_size_dev);
  hipDeviceSynchronize();
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "PrepareReverseGraph kernel failed." << endl;
    exit(-1);
  }
  ShrinkGraph<<<grid_size, block_size>>>(
      graph_new_dev, newg_list_size_dev, newg_revlist_size_dev, graph_old_dev,
      oldg_list_size_dev, oldg_revlist_size_dev);
  hipDeviceSynchronize();
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "ShrinkGraph kernel failed." << endl;
    exit(-1);
  }
  auto end = chrono::steady_clock::now();
  if (VERBOSE) {
    cerr << "Prepare kernel costs: "
         << (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
                    .count() /
                1e6
         << endl;
  }
}

void ToDevKNNGraph(NNDElement *dev_graph, vector<vector<NNDElement>> host_graph,
                   const int k) {
  NNDElement *host_graph_tmp = new NNDElement[host_graph.size() * k];
  for (int i = 0; i < host_graph.size(); i++) {
    memcpy(&host_graph_tmp[i * k], host_graph[i].data(),
           (size_t)k * sizeof(NNDElement));
  }
  hipMemcpy(dev_graph, host_graph_tmp,
             (size_t)host_graph.size() * k * sizeof(NNDElement),
             hipMemcpyHostToDevice);
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "ToDevKNNGraph failed." << endl;
    exit(-1);
  }
  delete[] host_graph_tmp;
}

void GetTestGraph(Graph *graph_new_ptr, Graph *graph_old_ptr,
                  int *graph_new_dev, int *newg_list_size_dev,
                  int *graph_old_dev, int *oldg_list_size_dev,
                  const int graph_size) {
  Graph &g_new = *graph_new_ptr;
  g_new.clear();
  g_new.resize(graph_size);
  Graph &g_old = *graph_old_ptr;
  g_old.clear();
  g_old.resize(graph_size);
  int *host_graph = new int[graph_size];
  int *newg_list_size = new int[graph_size];
  int *oldg_list_size = new int[graph_size];
  hipMemcpy(newg_list_size, newg_list_size_dev,
             (size_t)graph_size * sizeof(int), hipMemcpyDeviceToHost);
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "1. Get test graph failed." << endl;
    exit(-1);
  }
  hipMemcpy(oldg_list_size, oldg_list_size_dev,
             (size_t)graph_size * sizeof(int), hipMemcpyDeviceToHost);
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "2. Get test graph failed." << endl;
    exit(-1);
  }
  int *graph_new = new int[graph_size * (SAMPLE_NUM * 2)];
  int *graph_old = new int[graph_size * (SAMPLE_NUM * 2)];
  hipMemcpy(graph_new, graph_new_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int),
             hipMemcpyDeviceToHost);
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "3. Get test graph failed." << endl;
    exit(-1);
  }
  hipMemcpy(graph_old, graph_old_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int),
             hipMemcpyDeviceToHost);
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "4. Get test graph failed." << endl;
    exit(-1);
  }
  for (int i = 0; i < graph_size; i++) {
    int list_base_pos = i * (SAMPLE_NUM * 2);
    for (int j = 0; j < newg_list_size[i]; j++) {
      g_new[i].push_back(graph_new[list_base_pos + j]);
    }
  }
  for (int i = 0; i < graph_size; i++) {
    int list_base_pos = i * (SAMPLE_NUM * 2);
    for (int j = 0; j < oldg_list_size[i]; j++) {
      g_old[i].push_back(graph_old[list_base_pos + j]);
    }
  }
  // for (int i = 0; i < graph_size; i++) {
  //   sort(g_new[i].begin(), g_new[i].end());
  //   g_new[i].erase(unique(g_new[i].begin(), g_new[i].end()), g_new[i].end());

  //   sort(g_old[i].begin(), g_old[i].end());
  //   g_old[i].erase(unique(g_old[i].begin(), g_old[i].end()), g_old[i].end());
  // }
  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Get test graph failed." << endl;
    exit(-1);
  }
  delete[] newg_list_size;
  delete[] oldg_list_size;
  delete[] graph_new;
  delete[] graph_old;
  delete[] host_graph;
}

__device__ __forceinline__ NNDElement
__shfl_down_sync(const int mask, NNDElement var, const int delta,
                 const int width = WARP_SIZE) {
  NNDElement res;
  res.distance_ = __shfl_down_sync(mask, var.distance_, delta, width);
  res.label_ = __shfl_down_sync(mask, var.label_, delta, width);
  return res;
}

__device__ __forceinline__ NNDElement
__shfl_up_sync(const int mask, NNDElement var, const int delta,
               const int width = WARP_SIZE) {
  NNDElement res;
  res.distance_ = __shfl_up_sync(mask, var.distance_, delta, width);
  res.label_ = __shfl_up_sync(mask, var.label_, delta, width);
  return res;
}

template <typename T>
__device__ __forceinline__ T Min(const T &a, const T &b) {
  return a < b ? a : b;
}

__device__ NNDElement GetMinElement(const int *neighbs_id, const int list_id,
                                    const int list_size, const float *distances,
                                    const int distances_num) {
  int head_pos = list_id * (list_id - 1) / 2;
  int tail_pos = (list_id + 1) * list_id / 2;
  int y_num = tail_pos - head_pos;

  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  NNDElement min_elem = NNDElement(1e10, LARGE_INT);

  int it_num = GetItNum(y_num, WARP_SIZE);
  for (int it = 0; it < it_num; it++) {
    NNDElement elem;
    elem.SetLabel(neighbs_id[it * WARP_SIZE + lane_id]);
    int current_pos = head_pos + it * WARP_SIZE + lane_id;
    if (current_pos < tail_pos) {
      elem.SetDistance(distances[current_pos]);
    } else {
      elem = NNDElement(1e10, LARGE_INT);
    }
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
      elem = Min(elem, __shfl_down_sync(FULL_MASK, elem, offset));
    if (lane_id == 0) {
      min_elem = Min(elem, min_elem);
    }
  }

  head_pos = list_id * (list_id + 3) / 2;  // 0   2   5   9   14
  for (int it = 0; it < 2; it++) {
    NNDElement elem;
    int no = it * WARP_SIZE + lane_id;
    elem.SetLabel(neighbs_id[no + list_id + 1]);
    int current_pos = head_pos + no * (no + list_id * 2 + 1) / 2;
    if (current_pos < distances_num) {
      elem.SetDistance(distances[current_pos]);
    } else {
      elem = NNDElement(1e10, LARGE_INT);
    }
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
      elem = Min(elem, __shfl_down_sync(FULL_MASK, elem, offset));
    if (lane_id == 0) {
      min_elem = Min(elem, min_elem);
    }
  }
  return min_elem;
}

__device__ NNDElement GetMinElement2(const int list_id, const int list_size,
                                     const int *old_neighbs, const int num_old,
                                     const float *distances,
                                     const int distances_num) {
  int head_pos = list_id * num_old;
  int y_num = num_old;
  int tail_pos = head_pos + num_old;

  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  NNDElement min_elem = NNDElement(1e10, LARGE_INT);

  int it_num = GetItNum(y_num, WARP_SIZE);
  for (int it = 0; it < it_num; it++) {
    NNDElement elem;
    int no = it * WARP_SIZE + lane_id;
    elem.SetLabel(old_neighbs[no]);
    int current_pos = head_pos + no;
    if (current_pos < tail_pos) {
      elem.SetDistance(distances[current_pos]);
    } else {
      elem = NNDElement(1e10, LARGE_INT);
    }
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
      elem = Min(elem, __shfl_down_sync(FULL_MASK, elem, offset));
    if (lane_id == 0) {
      min_elem = Min(elem, min_elem);
    }
  }
  return min_elem;
}

__device__ NNDElement GetMinElement3(const int list_id, const int list_size,
                                     const int *new_neighbs, const int num_new,
                                     const int *old_neighbs, const int num_old,
                                     const float *distances,
                                     const int distances_num,
                                     const float *vectors) {
  int head_pos = list_id - num_new;
  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  NNDElement min_elem = NNDElement(1e10, LARGE_INT);

  int it_num = GetItNum(num_new, WARP_SIZE);
  for (int it = 0; it < it_num; it++) {
    NNDElement elem;
    int no = it * WARP_SIZE + lane_id;
    elem.SetLabel(new_neighbs[no]);
    int current_pos = head_pos + no * num_old;
    if (current_pos < distances_num) {
      elem.SetDistance(distances[current_pos]);
    } else {
      elem = NNDElement(1e10, LARGE_INT);
    }
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
      elem = Min(elem, __shfl_down_sync(FULL_MASK, elem, offset));
    if (lane_id == 0) {
      min_elem = Min(elem, min_elem);
    }
  }
  return min_elem;
}

__device__ uint GetNthSetBitPos(uint mask, int nth) {
  uint res;
  asm("fns.b32 %0,%1,%2,%3;" : "=r"(res) : "r"(mask), "r"(0), "r"(nth));
  return res;
}

__device__ void InsertToGlobalGraph(NNDElement elem, const int local_id,
                                    const int global_id,
                                    NNDElement *global_knn_graph,
                                    int *global_locks) {
  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  int global_pos_base = global_id * NEIGHB_NUM_PER_LIST;
  elem.distance_ = __shfl_sync(FULL_MASK, elem.distance_, 0);
  elem.label_ = __shfl_sync(FULL_MASK, elem.label_, 0);
  int loop_flag = 0;
  do {
    int block_id = elem.label() % NEIGHB_BLOCKS_NUM;
    if (lane_id == 0)
      loop_flag =
          atomicCAS(&global_locks[global_id * NEIGHB_BLOCKS_NUM + block_id], 0,
                    1) == 0;
    loop_flag = __shfl_sync(FULL_MASK, loop_flag, 0);
    if (loop_flag == 1) {
      NNDElement knn_list_frag;
      int local_pos = block_id * WARP_SIZE + lane_id;
      int global_pos = global_pos_base + local_pos;
      if (local_pos < NEIGHB_NUM_PER_LIST)
        knn_list_frag = global_knn_graph[global_pos];
      else
        knn_list_frag = NNDElement(1e10, LARGE_INT);

      int pos_to_insert = -1;
      NNDElement prev_elem = __shfl_up_sync(FULL_MASK, knn_list_frag, 1);
      if (lane_id == 0) prev_elem = NNDElement(-1e10, -LARGE_INT);
      if (elem > prev_elem && elem < knn_list_frag)
        pos_to_insert = block_id * WARP_SIZE + lane_id;
      else if (elem == prev_elem || elem == knn_list_frag)
        pos_to_insert = -2;
      uint mask = __ballot_sync(FULL_MASK, pos_to_insert >= 0);
      if (mask) {
        uint set_lane_id = GetNthSetBitPos(mask, 1);
        pos_to_insert = __shfl_sync(FULL_MASK, pos_to_insert, set_lane_id);
        // assert(false);
      }
      if (pos_to_insert >= 0) {
        int local_pos = block_id * WARP_SIZE + lane_id;
        if (local_pos > pos_to_insert) {
          local_pos++;
        } else if (local_pos == pos_to_insert) {
          global_knn_graph[global_pos_base + local_pos] = elem;
          local_pos++;
        }
        int global_pos = global_pos_base + local_pos;
        if (local_pos < (block_id + 1) * WARP_SIZE &&
            local_pos < NEIGHB_NUM_PER_LIST)
          global_knn_graph[global_pos] = knn_list_frag;
      }
    }
    __threadfence();
    if (loop_flag && lane_id == 0) {
      atomicExch(&global_locks[global_id * NEIGHB_BLOCKS_NUM + block_id], 0);
    }
    __nanosleep(32);
  } while (!loop_flag);
}

__global__ void NewNeighborsCompareKernel(
    NNDElement *knn_graph, int *global_locks, const float *vectors,
    const int *graph_new, const int *size_new, const int num_new_max) {
  extern __shared__ char buffer[];

  __shared__ float *shared_vectors, *distances;
  __shared__ int *neighbors;
  __shared__ int gnew_base_pos, num_new;

  int tx = threadIdx.x;
  if (tx == 0) {
    shared_vectors = (float *)buffer;
    size_t offset = num_new_max * SKEW_DIM * sizeof(float);
    distances = (float *)((char *)buffer + offset);
    neighbors = (int *)((char *)distances +
                        (num_new_max * (num_new_max - 1) / 2) * sizeof(float));
  }
  __syncthreads();

  int list_id = blockIdx.x;
  int block_dim_x = blockDim.x;

  if (tx == 0) {
    gnew_base_pos = list_id * (SAMPLE_NUM * 2);
  } else if (tx == 32) {
    num_new = size_new[list_id];
  }
  __syncthreads();
  int neighb_num = num_new;
  if (tx < neighb_num) {
    neighbors[tx] = graph_new[gnew_base_pos + tx];
  }
  __syncthreads();
  int num_vec_per_it = block_dim_x / VEC_DIM;
  int num_it = GetItNum(neighb_num, num_vec_per_it);
  for (int i = 0; i < num_it; i++) {
    int x = i * num_vec_per_it + tx / VEC_DIM;
    if (x >= neighb_num) continue;
    int y = tx % VEC_DIM;
    int vec_id = neighbors[x];
    shared_vectors[x * SKEW_DIM + y] = vectors[vec_id * VEC_DIM + y];
  }
  __syncthreads();

  int calc_num = (neighb_num * (neighb_num - 1)) / 2;

  num_it = GetItNum(calc_num, block_dim_x);
  for (int i = 0; i < num_it; i++) {
    int no = i * block_dim_x + tx;
    if (no >= calc_num) continue;
    int idx = no + 1;
    int x = ceil(sqrt(2 * idx + 0.25) - 0.5);
    if (x >= neighb_num) continue;
    int y = idx - (x - 1) * x / 2 - 1;
    if (y >= neighb_num) continue;
    float sum = 0;
    int base_x = x * SKEW_DIM;
    int base_y = y * SKEW_DIM;
    for (int j = 0; j < VEC_DIM; j++) {
      float diff = shared_vectors[base_x + j] - shared_vectors[base_y + j];
      sum += diff * diff;
    }
    distances[no] = sum;
  }
  __syncthreads();
  // num_it = GetItNum(NEIGHB_NUM_PER_LIST, NEIGHB_CACHE_NUM);

  int list_size = NEIGHB_CACHE_NUM;
  int num_it3 = GetItNum(neighb_num, block_dim_x / WARP_SIZE);
  for (int j = 0; j < num_it3; j++) {
    int list_id = j * (block_dim_x / WARP_SIZE) + tx / WARP_SIZE;
    if (list_id >= neighb_num) continue;
    NNDElement min_elem =
        GetMinElement(neighbors, list_id, list_size, distances, calc_num);
    InsertToGlobalGraph(min_elem, list_id, neighbors[list_id], knn_graph,
                        global_locks);
  }
}

// blockDim.x = TILE_WIDTH * TILE_WIDTH;
__device__ void GetNewOldDistancesTiled(float *distances, const float *vectors,
                                        const int *new_neighbors,
                                        const int num_new,
                                        const int *old_neighbors,
                                        const int num_old) {
  __shared__ float nsv[TILE_WIDTH][SKEW_TILE_WIDTH];  // New shared vectors
  __shared__ float osv[TILE_WIDTH][SKEW_TILE_WIDTH];  // Old shared vectors
  const int width = VEC_DIM;

  int tx = threadIdx.x;
  int t_row = tx / TILE_WIDTH;
  int t_col = tx % TILE_WIDTH;
  int row_num = (int)(ceil(1.0 * num_new / TILE_WIDTH));
  int col_num = (int)(ceil(1.0 * num_old / TILE_WIDTH));
  int tiles_num = row_num * col_num;
  // if (threadIdx.x == 0) {
  //     printf("%d %d %d\n", row_num, col_num, tiles_num);
  // }
  for (int i = 0; i < tiles_num; i++) {
    float distance = -1.0;
    int row_new = i / col_num * TILE_WIDTH;
    int row_old = i % col_num * TILE_WIDTH;
    // Assume that the dimension of vectors larger than num of neighbors.
    for (int ph = 0; ph < ceil(width / (float)TILE_WIDTH); ph++) {
      if ((row_new + t_row < num_new) && (ph * TILE_WIDTH + t_col < VEC_DIM)) {
        nsv[t_row][t_col] = vectors[new_neighbors[row_new + t_row] * VEC_DIM +
                                    ph * TILE_WIDTH + t_col];
      } else {
        nsv[t_row][t_col] = 1e10;
      }

      if ((row_old + t_col < num_old) && (ph * TILE_WIDTH + t_row < VEC_DIM)) {
        osv[t_col][t_row] = vectors[old_neighbors[row_old + t_col] * VEC_DIM +
                                    ph * TILE_WIDTH + t_row];
      } else {
        osv[t_col][t_row] = 1e10;
      }
      __syncthreads();

      for (int k = 0; k < TILE_WIDTH; k++) {
        float a = nsv[t_row][k], b = osv[t_col][k];
        if (a > 1e9 || b > 1e9) {
        } else {
          float diff = a - b;
          if (distance == -1.0) distance = 0;
          distance += diff * diff;
        }
      }
      __syncthreads();
    }
    if (distance != -1.0) {
      distances[(row_new + t_row) * num_old + row_old + t_col] = distance;
    }
  }
}

__global__ void TiledNewOldNeighborsCompareKernel(
    NNDElement *knn_graph, int *global_locks, const float *vectors,
    const int *graph_new, const int *size_new, const int num_new_max,
    const int *graph_old, const int *size_old, const int num_old_max) {
  extern __shared__ char buffer[];

  __shared__ float *distances;
  __shared__ int *neighbors;

  __shared__ int gnew_base_pos, gold_base_pos, num_new, num_old;

  int tx = threadIdx.x;
  if (tx == 0) {
    distances = (float *)buffer;
    neighbors = (int *)((char *)distances +
                        (num_new_max * num_old_max) * sizeof(float));
  }
  __syncthreads();

  int list_id = blockIdx.x;
  int block_dim_x = blockDim.x;

  if (tx == 0) {
    gnew_base_pos = list_id * (SAMPLE_NUM * 2);
    gold_base_pos = list_id * (SAMPLE_NUM * 2);
  } else if (tx == 32) {
    num_new = size_new[list_id];
    num_old = size_old[list_id];
  }
  __syncthreads();
  int neighb_num = num_new + num_old;
  if (tx < num_new) {
    neighbors[tx] = graph_new[gnew_base_pos + tx];
  } else if (tx >= num_new && tx < neighb_num) {
    neighbors[tx] = graph_old[gnew_base_pos + tx - num_new];
  }
  __syncthreads();

  GetNewOldDistancesTiled(distances, vectors, neighbors, num_new,
                          neighbors + num_new, num_old);
  __syncthreads();

  int calc_num = num_new * num_old;
  // int num_it = GetItNum(NEIGHB_NUM_PER_LIST, NEIGHB_CACHE_NUM);

  // Read list to cache
  int list_size = NEIGHB_CACHE_NUM;
  int num_it3 = GetItNum(neighb_num, block_dim_x / WARP_SIZE);
  for (int j = 0; j < num_it3; j++) {
    int list_id = j * (block_dim_x / WARP_SIZE) + tx / WARP_SIZE;
    if (list_id >= neighb_num) continue;
    NNDElement min_elem(1e10, LARGE_INT);
    if (list_id < num_new) {
      min_elem =
          Min(min_elem, GetMinElement2(list_id, list_size, neighbors + num_new,
                                       num_old, distances, calc_num));
    } else {
      min_elem =
          Min(min_elem, GetMinElement3(list_id, list_size, neighbors, num_new,
                                       neighbors + num_new, num_old, distances,
                                       calc_num, vectors));
    }
    InsertToGlobalGraph(min_elem, list_id, neighbors[list_id], knn_graph,
                        global_locks);
  }
  __syncthreads();
}

__global__ void MarkAllToOld(NNDElement *knn_graph) {
  int list_id = blockIdx.x;
  int tx = threadIdx.x;
  int graph_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  knn_graph[graph_base_pos + tx].MarkOld();
}

pair<int *, int *> ReadGraphToGlobalMemory(const Graph &graph) {
  int pos = 0;
  vector<int> edges, dest;
  for (int i = 0; i < graph.size(); i++) {
    edges.push_back(pos);
    // dest.push_back(graph[i].size());
    // pos++;
    for (int j = 0; j < graph[i].size(); j++) {
      dest.push_back(graph[i][j]);
      pos++;
    }
  }
  edges.push_back(pos);

  int *edges_dev, *dest_dev;
  hipError_t cuda_status0, cuda_status1;
  cuda_status0 = hipMalloc(&edges_dev, edges.size() * sizeof(int));
  cuda_status1 = hipMalloc(&dest_dev, dest.size() * sizeof(int));
  if (cuda_status0 != hipSuccess || cuda_status1 != hipSuccess) {
    cerr << "CudaMalloc failed" << endl;
    exit(-1);
  }

  cuda_status0 = hipMemcpy(edges_dev, edges.data(), edges.size() * sizeof(int),
                            hipMemcpyHostToDevice);
  cuda_status1 = hipMemcpy(dest_dev, dest.data(), dest.size() * sizeof(int),
                            hipMemcpyHostToDevice);
  if (cuda_status0 != hipSuccess || cuda_status1 != hipSuccess) {
    cerr << "CudaMemcpy failed" << endl;
    exit(-1);
  }
  return make_pair(edges_dev, dest_dev);
}

__global__ void TestKernel(NNDElement *knn_graph) {
  for (int i = 0; i < 10000 * 30; i++) {
    if (knn_graph[i].distance() == 0 && knn_graph[i].label() == 0) {
      printf("check %d %f\n", i, knn_graph[i].distance());
    }
  }
  return;
}

NNDElement *ReadKNNGraphToGlobalMemory(
    const vector<vector<NNDElement>> &knn_graph) {
  int k = knn_graph[0].size();
  NNDElement *knn_graph_dev;
  NNDElement *knn_graph_host = new NNDElement[knn_graph.size() * k];
  int idx = 0;
  for (int i = 0; i < knn_graph.size(); i++) {
    for (int j = 0; j < k; j++) {
      const auto &item = knn_graph[i][j];
      knn_graph_host[idx++] = item;
    }
  }

  auto cuda_status =
      hipMalloc(&knn_graph_dev, knn_graph.size() * k * sizeof(NNDElement));
  if (cuda_status != hipSuccess) {
    cerr << "knn_graph hipMalloc failed." << endl;
    exit(-1);
  }
  cuda_status = hipMemcpy(knn_graph_dev, knn_graph_host,
                           knn_graph.size() * k * sizeof(NNDElement),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "knn_graph hipMemcpyHostToDevice failed." << endl;
    exit(-1);
  }
  delete[] knn_graph_host;
  return knn_graph_dev;
}

int GetMaxListSize(const Graph &g) {
  int res = 0;
  for (const auto &list : g) {
    res = max((int)list.size(), res);
  }
  return res;
}

int GetMaxListSize(int *list_size_dev, const int g_size) {
  thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(list_size_dev);
  return *thrust::max_element(dev_ptr, dev_ptr + g_size);
}

float UpdateGraph(NNDElement *origin_knn_graph_dev, const size_t g_size,
                  const float *vectors_dev, int *newg_dev,
                  int *newg_list_size_dev, int *oldg_dev,
                  int *oldg_list_size_dev, const int k,
                  const bool calc_between_new_neighbs = true) {
  float kernel_time = 0;
  hipError_t cuda_status;

  int *global_locks_dev;
  hipMalloc(&global_locks_dev, g_size * sizeof(int) * NEIGHB_BLOCKS_NUM);
  hipMemset(global_locks_dev, 0, g_size * sizeof(int) * NEIGHB_BLOCKS_NUM);
  cuda_status = hipGetLastError();

  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Initiate failed" << endl;
    exit(-1);
  }

  dim3 block_size(640);
  dim3 grid_size(g_size);
  // cerr << "Start kernel." << endl;
  const int num_new_max = GetMaxListSize(newg_list_size_dev, g_size);
  const int num_old_max = GetMaxListSize(oldg_list_size_dev, g_size);
  if (VERBOSE) {
    cerr << "Num new max: " << num_new_max << endl;
    cerr << "Num old max: " << num_old_max << endl;
  }
  size_t shared_memory_size =
      num_new_max * SKEW_DIM * sizeof(float) +
      (num_new_max * (num_new_max - 1) / 2) * sizeof(float) +
      num_new_max * sizeof(int);

  if (VERBOSE) {
    cerr << "Shmem kernel1 costs: " << shared_memory_size << endl;
  }

  auto start = chrono::steady_clock::now();
  MarkAllToOld<<<g_size, NEIGHB_NUM_PER_LIST>>>(origin_knn_graph_dev);
  if (calc_between_new_neighbs) {
    NewNeighborsCompareKernel<<<grid_size, block_size, shared_memory_size>>>(
        origin_knn_graph_dev, global_locks_dev, vectors_dev, newg_dev,
        newg_list_size_dev, num_new_max);
  }
  int neighb_num_max = num_new_max + num_old_max;
  block_size = dim3(TILE_WIDTH * TILE_WIDTH);
  shared_memory_size = (num_new_max * num_old_max) * sizeof(float) +
                       neighb_num_max * sizeof(int);
  if (VERBOSE) {
    cerr << "Shmem tiled kernel2 costs: " << shared_memory_size << endl;
  }
  TiledNewOldNeighborsCompareKernel<<<grid_size, block_size,
                                      shared_memory_size>>>(
      origin_knn_graph_dev, global_locks_dev, vectors_dev, newg_dev,
      newg_list_size_dev, num_new_max, oldg_dev, oldg_list_size_dev,
      num_old_max);
  hipDeviceSynchronize();
  auto end = chrono::steady_clock::now();
  kernel_time =
      (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
          .count() /
      1e6;
  cuda_status = hipGetLastError();

  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Kernel failed" << endl;
    exit(-1);
  }
  // cerr << "End kernel." << endl;
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "knn_graph hipMemcpy failed" << endl;
    exit(-1);
  }

  hipFree(global_locks_dev);
  return kernel_time;
}

void OutputGraph(const xmuknn::Graph &g, const string &path) {
  ofstream out(path);
  for (int i = 0; i < g.size(); i++) {
    out << g[i].size() << "\t";
    for (int j = 0; j < g[i].size(); j++) {
      out << g[i][j] << "\t";
    }
    out << endl;
  }
  out.close();
}

void OutputGraph(const vector<vector<NNDElement>> &g, const string &path) {
  ofstream out(path);
  for (int i = 0; i < g.size(); i++) {
    out << g[i].size() << "\t";
    for (int j = 0; j < g[i].size(); j++) {
      out << g[i][j].label() << "\t";
    }
    out << endl;
  }
  out.close();
}

__global__ void InitKNNGraphIndexKernel(
    NNDElement *knn_graph, const int graph_size,
    const unsigned long long *random_sequence = 0) {
  int list_id = blockIdx.x;
  int tx = threadIdx.x;
  int pos = list_id * NEIGHB_NUM_PER_LIST + tx;
  int block_id = tx / WARP_SIZE;

  knn_graph[pos].SetDistance(1e10);
  int label;
  if (random_sequence == 0) {
    label = xorshift64star(pos) % (unsigned long long)graph_size;
  } else {
    label = random_sequence[pos] % (unsigned long long)graph_size;
  }
  while (label % NEIGHB_BLOCKS_NUM != block_id || label == list_id) {
    label = (label + 1) % (unsigned long long)graph_size;
  }
  knn_graph[pos].SetLabel(label);
}

__global__ void InitKNNGraphDistanceKernel(NNDElement *knn_graph,
                                           const int graph_size,
                                           const float *vectors) {
  int list_id = blockIdx.x;
  int tx = threadIdx.x;
  int lane_id = threadIdx.x % WARP_SIZE;
  int vec_a_pos = list_id * VEC_DIM;

  for (int i = 0; i < NEIGHB_NUM_PER_LIST; i++) {
    int it_num = GetItNum(VEC_DIM, blockDim.x);
    float sum = 0;
    int vec_b_pos =
        knn_graph[list_id * NEIGHB_NUM_PER_LIST + i].label() * VEC_DIM;
    for (int j = 0; j < it_num; j++) {
      int vec_elem_pos = j * blockDim.x + tx;
      float elem_a, elem_b, diff;
      if (vec_elem_pos < VEC_DIM) {
        elem_a = vectors[vec_a_pos + vec_elem_pos];
        elem_b = vectors[vec_b_pos + vec_elem_pos];
        diff = elem_a - elem_b;
        diff *= diff;
      } else {
        diff = 0;
      }
      for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
        diff = diff + __shfl_down_sync(FULL_MASK, diff, offset);
      sum += diff;
    }
    if (lane_id == 0) {
      knn_graph[list_id * NEIGHB_NUM_PER_LIST + i].SetDistance(sum);
    }
  }
}

__global__ void SortEachBlockKernel(NNDElement *knn_graph,
                                    const int graph_size) {
  int list_id = blockIdx.x;
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  int global_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  int block_id = tx / WARP_SIZE;

  NNDElement elem;
  int pos = block_id * WARP_SIZE + lane_id;
  if (pos < NEIGHB_NUM_PER_LIST) {
    elem = knn_graph[global_base_pos + pos];
  } else {
    elem.SetDistance(1e10);
    elem.SetLabel(LARGE_INT);
  }
  BitonicSort(&elem, lane_id);
  if (pos < NEIGHB_NUM_PER_LIST) {
    knn_graph[global_base_pos + pos] = elem;
  }
  return;
}

__global__ void SortKNNGraphKernel(NNDElement *knn_graph,
                                   const int graph_size) {
  __shared__ NNDElement knn_list_cache[NEIGHB_NUM_PER_LIST];
  __shared__ NNDElement sorted_elements_cache[32];
  __shared__ NNDElement merged_list_cache[NEIGHB_NUM_PER_LIST];

  int list_id = blockIdx.x;
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  int global_base_pos = list_id * NEIGHB_NUM_PER_LIST;
  int tx = threadIdx.x;
  int lane_id = tx % WARP_SIZE;
  int list_size = 0;
  for (int i = 0; i < it_num; i++) {
    int pos = i * WARP_SIZE + tx;
    if (pos >= NEIGHB_NUM_PER_LIST) break;
    merged_list_cache[pos] = knn_list_cache[pos] = NNDElement(1e10, LARGE_INT);
  }
  for (int i = 0; i < it_num; i++) {
    NNDElement elem;
    int pos = i * WARP_SIZE + tx;
    if (pos >= NEIGHB_NUM_PER_LIST) {
      elem.SetDistance(1e10);
      elem.SetLabel(LARGE_INT);
    } else {
      elem = knn_graph[global_base_pos + pos];
    }
    BitonicSort(&elem, lane_id);
    sorted_elements_cache[lane_id] = elem;
    if (lane_id == 0) {
      list_size = MergeList(knn_list_cache, list_size, sorted_elements_cache,
                            WARP_SIZE, merged_list_cache);
    }
    list_size = __shfl_sync(FULL_MASK, list_size, 0);
    int copy_it_num = GetItNum(list_size, WARP_SIZE);
    for (int j = 0; j < copy_it_num; j++) {
      int pos = j * WARP_SIZE + lane_id;
      if (pos >= NEIGHB_NUM_PER_LIST) break;
      knn_list_cache[pos] = merged_list_cache[pos];
    }
  }
  __syncthreads();
  for (int i = 0; i < it_num; i++) {
    int pos = i * WARP_SIZE + tx;
    if (pos >= NEIGHB_NUM_PER_LIST) break;
    knn_graph[global_base_pos + pos] = knn_list_cache[pos];
  }
}

void InitRandomKNNGraph(NNDElement *knn_graph_dev, const int graph_size,
                        const float *vectors_dev,
                        bool start_from_random_index = true,
                        bool using_thrust_random = true) {
  auto start = chrono::steady_clock().now();
  if (start_from_random_index) {
    if (using_thrust_random) {
      thrust::device_vector<unsigned long long> dev_random_sequence(
          graph_size * NEIGHB_NUM_PER_LIST);
      DevRNGLongLong(thrust::raw_pointer_cast(dev_random_sequence.data()),
                     graph_size * NEIGHB_NUM_PER_LIST);
      InitKNNGraphIndexKernel<<<graph_size, NEIGHB_NUM_PER_LIST>>>(
          knn_graph_dev, graph_size,
          thrust::raw_pointer_cast(dev_random_sequence.data()));
      hipDeviceSynchronize();
    } else {
      InitKNNGraphIndexKernel<<<graph_size, NEIGHB_NUM_PER_LIST>>>(
          knn_graph_dev, graph_size);
      hipDeviceSynchronize();
      auto cuda_status = hipGetLastError();
      if (cuda_status != hipSuccess) {
        cerr << hipGetErrorString(cuda_status) << endl;
        cerr << "InitKNNGraphIndexKernel failed." << endl;
        exit(-1);
      }
    }
  }

  // vector<vector<NNDElement>> g(graph_size);
  // #pragma omp parallel for
  // for (int i = 0; i < graph_size; i++) {
  //   vector<int> exclusion = {i};
  //   vector<int> tmp_vec;
  //   xmuknn::GenerateRandomSequence(tmp_vec, NEIGHB_NUM_PER_LIST, graph_size,
  //                                  exclusion);
  //   for (int j = 0; j < NEIGHB_NUM_PER_LIST; j++) {
  //     int nb_id = tmp_vec[j];
  //     g[i].emplace_back(1e10, nb_id);
  //   }
  // }
  // ToDevKNNGraph(knn_graph_dev, g, NEIGHB_NUM_PER_LIST);

  InitKNNGraphDistanceKernel<<<graph_size, 32>>>(knn_graph_dev, graph_size,
                                                 vectors_dev);
  hipDeviceSynchronize();
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "InitKNNGraphDistanceKernel failed." << endl;
    exit(-1);
  }
  SortEachBlockKernel<<<graph_size, WARP_SIZE * NEIGHB_BLOCKS_NUM>>>(
      knn_graph_dev, graph_size);
  // SortKNNGraphKernel<<<graph_size, 32>>>(knn_graph_dev, graph_size);
  hipDeviceSynchronize();
  auto end = chrono::steady_clock().now();

  cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "SortKNNGraphKernel failed." << endl;
    exit(-1);
  }
  if (VERBOSE) {
    cerr << "Initiate costs: "
         << (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
                    .count() /
                1e6
         << endl;
  }
}

__global__ void MergeBlocksKernel(NNDElement *knn_graph, const int graph_size) {
  __shared__ NNDElement c_cache[NEIGHB_NUM_PER_LIST];
  __shared__ NNDElement tmp_cache[NEIGHB_NUM_PER_LIST];
  int list_id = blockIdx.x;
  int global_pos_base = list_id * NEIGHB_NUM_PER_LIST;
  int tx = threadIdx.x;
  int list_size = 0;
  c_cache[tx] = knn_graph[global_pos_base + tx];
  list_size = WARP_SIZE;
  for (int i = 1; i < NEIGHB_BLOCKS_NUM; i++) {
    list_size = MergeList(c_cache, list_size,
                          &knn_graph[global_pos_base + i * WARP_SIZE],
                          WARP_SIZE, tmp_cache);
    int it_num = GetItNum(list_size, WARP_SIZE);
    for (int j = 0; j < it_num; j++) {
      int pos = j * WARP_SIZE + tx;
      if (pos < list_size) {
        c_cache[pos] = tmp_cache[pos];
      }
    }
  }
  int it_num = GetItNum(NEIGHB_NUM_PER_LIST, WARP_SIZE);
  for (int i = 0; i < it_num; i++) {
    int pos = i * WARP_SIZE + tx;
    if (pos < NEIGHB_NUM_PER_LIST) {
      knn_graph[global_pos_base + pos] = c_cache[pos];
    }
  }
}

void MergeBlocksInNNLists(NNDElement *knn_graph, const int graph_size) {
  MergeBlocksKernel<<<graph_size, WARP_SIZE>>>(knn_graph, graph_size);
  hipDeviceSynchronize();
}

namespace gpuknn {
void NNDescentForMerge(NNDElement *knngraph_dev, const float *vectors_dev,
                       const int vecs_size, const int vecs_dim,
                       const int split_pos, const int iteration) {
  int k = NEIGHB_NUM_PER_LIST;
  int *graph_new_dev, *newg_list_size_dev, *graph_old_dev, *oldg_list_size_dev;
  int *newg_revlist_size_dev, *oldg_revlist_size_dev;
  int graph_size = vecs_size;
  hipMalloc(&graph_new_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int));
  hipMalloc(&newg_list_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&newg_revlist_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&graph_old_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int));
  hipMalloc(&oldg_list_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&oldg_revlist_size_dev, (size_t)graph_size * sizeof(int));
  Graph result(vecs_size);
  InitRandomKNNGraph(knngraph_dev, graph_size, vectors_dev, false, false);
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Init failed" << endl;
    exit(-1);
  }
  float iteration_costs = 0;
  Graph newg, oldg;
  float get_nb_graph_time = 0;
  float kernel_costs = 0;
  auto sum_start = chrono::steady_clock::now();
  long long cmp_times = 0;
  for (int t = 0; t < iteration; t++) {
    if (VERBOSE) {
      cerr << "Start generating NBGraph." << endl;
    }
    // Should be removed after testing.
    auto start = chrono::steady_clock::now();
    PrepareForUpdateForMerge(graph_new_dev, newg_list_size_dev,
                             newg_revlist_size_dev, graph_old_dev,
                             oldg_list_size_dev, oldg_revlist_size_dev,
                             knngraph_dev, split_pos, graph_size);
    auto end = chrono::steady_clock::now();
    float tmp_time =
        (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count() /
        1e6;
    get_nb_graph_time += tmp_time;
    if (VERBOSE) {
      cerr << "GetNBGraph costs " << tmp_time << endl;
    }
    start = chrono::steady_clock::now();
    float tmp_kernel_costs = UpdateGraph(
        knngraph_dev, graph_size, vectors_dev, graph_new_dev,
        newg_list_size_dev, graph_old_dev, oldg_list_size_dev, k, false);
    kernel_costs += tmp_kernel_costs;
    end = chrono::steady_clock::now();
    float it_tmp_costs =
        (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count() /
        1e6;
    iteration_costs += it_tmp_costs;
    if (VERBOSE) {
      cerr << "Kernel costs " << tmp_kernel_costs << endl;
      cerr << endl;
    }
  }
  MergeBlocksInNNLists(knngraph_dev, graph_size);
  auto sum_end = chrono::steady_clock::now();
  float sum_costs = (float)chrono::duration_cast<std::chrono::microseconds>(
                        sum_end - sum_start)
                        .count() /
                    1e6;
  // sift10k in cpu should be 0.6s;
  if (VERBOSE) {
    cerr << "Compare times: " << cmp_times << endl;
    cerr << "FLOPS: " << cmp_times * 128 * 3 / kernel_costs / pow(1024.0, 3)
         << "G" << endl;
    cerr << "Kernel costs: " << kernel_costs << endl;
    cerr << "Update costs: " << iteration_costs << endl;
    cerr << "Get NB graph costs: " << get_nb_graph_time << endl;
    cerr << "All procedure costs: " << sum_costs << endl;
    cerr << endl;
  }
  hipFree(graph_new_dev);
  hipFree(graph_old_dev);

  hipFree(newg_list_size_dev);
  hipFree(oldg_list_size_dev);
  hipFree(newg_revlist_size_dev);
  hipFree(oldg_revlist_size_dev);
}
void NNDescent(NNDElement **knngraph_result_ptr, const float *vectors_dev,
               const int vecs_size, const int vecs_dim, const int iteration,
               const bool store_result_in_device) {
  int k = NEIGHB_NUM_PER_LIST;
  int *graph_new_dev, *newg_list_size_dev, *graph_old_dev, *oldg_list_size_dev;
  int *newg_revlist_size_dev, *oldg_revlist_size_dev;
  int graph_size = vecs_size;
  NNDElement *&knngraph_result = *knngraph_result_ptr;
  hipMalloc(&graph_new_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int));
  hipMalloc(&newg_list_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&newg_revlist_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&graph_old_dev,
             (size_t)graph_size * (SAMPLE_NUM * 2) * sizeof(int));
  hipMalloc(&oldg_list_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&oldg_revlist_size_dev, (size_t)graph_size * sizeof(int));
  hipMalloc(&knngraph_result, (size_t)graph_size * k * sizeof(NNDElement));
  Graph result(vecs_size);
  vector<vector<NNDElement>> g(vecs_size);
  // InitRandomKNNGraph(knngraph_result, graph_size, vectors_dev);
  InitRandomKNNGraph(knngraph_result, graph_size, vectors_dev, true, false);
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << hipGetErrorString(cuda_status) << endl;
    cerr << "Init failed" << endl;
    exit(-1);
  }

  float iteration_costs = 0;
  Graph newg, oldg;
  float get_nb_graph_time = 0;
  float kernel_costs = 0;
  auto sum_start = chrono::steady_clock::now();
  long long cmp_times = 0;
  for (int t = 0; t < iteration; t++) {
    if (VERBOSE) {
      cerr << "Start generating NBGraph." << endl;
    }
    // Should be removed after testing.
    auto start = chrono::steady_clock::now();
    PrepareForUpdate(graph_new_dev, newg_list_size_dev, newg_revlist_size_dev,
                     graph_old_dev, oldg_list_size_dev, oldg_revlist_size_dev,
                     knngraph_result, graph_size);
    auto end = chrono::steady_clock::now();
    float tmp_time =
        (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count() /
        1e6;
    get_nb_graph_time += tmp_time;
    if (VERBOSE) {
      cerr << "GetNBGraph costs " << tmp_time << endl;
    }
    start = chrono::steady_clock::now();
    float tmp_kernel_costs =
        UpdateGraph(knngraph_result, graph_size, vectors_dev, graph_new_dev,
                    newg_list_size_dev, graph_old_dev, oldg_list_size_dev, k);
    kernel_costs += tmp_kernel_costs;
    end = chrono::steady_clock::now();
    float it_tmp_costs =
        (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count() /
        1e6;
    iteration_costs += it_tmp_costs;
    if (VERBOSE) {
      cerr << "Kernel costs " << tmp_kernel_costs << endl;
      cerr << endl;
    }
  }
  MergeBlocksInNNLists(knngraph_result, graph_size);
  auto sum_end = chrono::steady_clock::now();
  float sum_costs = (float)chrono::duration_cast<std::chrono::microseconds>(
                        sum_end - sum_start)
                        .count() /
                    1e6;
  // sift10k in cpu should be 0.6s;
  if (VERBOSE) {
    cerr << "Compare times: " << cmp_times << endl;
    cerr << "FLOPS: " << cmp_times * 128 * 3 / kernel_costs / pow(1024.0, 3)
         << "G" << endl;
    cerr << "Kernel costs: " << kernel_costs << endl;
    cerr << "Update costs: " << iteration_costs << endl;
    cerr << "Get NB graph costs: " << get_nb_graph_time << endl;
    cerr << "All procedure costs: " << sum_costs << endl;
    cerr << endl;
  }
  hipFree(graph_new_dev);
  hipFree(graph_old_dev);

  hipFree(newg_list_size_dev);
  hipFree(oldg_list_size_dev);
  hipFree(newg_revlist_size_dev);
  hipFree(oldg_revlist_size_dev);

  if (!store_result_in_device) {
    NNDElement *host_graph;
    ToHostKNNGraph(&host_graph, knngraph_result, graph_size,
                   NEIGHB_NUM_PER_LIST);
    hipFree(knngraph_result);
    knngraph_result = host_graph;
  }
}
vector<vector<NNDElement>> NNDescent(const float *vectors, const int vecs_size,
                                     const int vecs_dim, const int iteration) {
  int k = NEIGHB_NUM_PER_LIST;
  auto cuda_status = hipSetDevice(DEVICE_ID);

  float *vectors_dev;
  hipMalloc(&vectors_dev, (size_t)vecs_size * vecs_dim * sizeof(float));
  hipMemcpy(vectors_dev, vectors, (size_t)vecs_size * vecs_dim * sizeof(float),
             hipMemcpyHostToDevice);
  NNDElement *knn_graph_dev;

  auto start = chrono::steady_clock::now();
  NNDescent(&knn_graph_dev, vectors_dev, vecs_size, vecs_dim, iteration);
  auto end = chrono::steady_clock::now();
  cerr << "GPU NNDescent costs: "
       << (float)chrono::duration_cast<std::chrono::microseconds>(end - start)
                  .count() /
              1e6
       << endl;
  vector<vector<NNDElement>> g;
  ToHostKNNGraph(&g, knn_graph_dev, vecs_size, k);  // 0.6 / 6.6
  hipFree(vectors_dev);
  hipFree(knn_graph_dev);
  return g;
}
}  // namespace gpuknn