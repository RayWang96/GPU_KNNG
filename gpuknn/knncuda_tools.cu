#include "hip/hip_runtime.h"
#include <assert.h>
#include <hiprand.h>

#include <chrono>
#include <fstream>
#include <iomanip>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "knncuda_tools.cuh"
using namespace std;
void DevRNGLongLong(unsigned long long *dev_data, int n) {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen,
                        hiprandRngType_t::HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
  hiprandSetPseudoRandomGeneratorSeed(gen, clock());
  hiprandGenerateLongLong(gen, dev_data, n);
}

__device__ int GetItNum(const int sum_num, const int num_per_it) {
  return sum_num / num_per_it + (sum_num % num_per_it != 0);
}

void ToHostKNNGraph(vector<vector<NNDElement>> *origin_knn_graph_ptr,
                    const NNDElement *knn_graph_dev, const int size,
                    const int neighb_num) {
  NNDElement *knn_graph = new NNDElement[size * neighb_num];
  hipMemcpy(knn_graph, knn_graph_dev,
             (size_t)size * neighb_num * sizeof(NNDElement),
             hipMemcpyDeviceToHost);
  auto &origin_knn_graph = *origin_knn_graph_ptr;
  vector<NNDElement> neighb_list;
  origin_knn_graph = vector<vector<NNDElement>>(size);
  for (int i = 0; i < size; i++) {
    neighb_list.clear();
    for (int j = 0; j < neighb_num; j++) {
      neighb_list.push_back(knn_graph[i * neighb_num + j]);
    }
    origin_knn_graph[i] = neighb_list;
  }
  delete[] knn_graph;
}

void ToHostKNNGraph(NNDElement **host_knn_graph_ptr,
                    const NNDElement *knn_graph_dev, const int size,
                    const int neighb_num) {
  NNDElement *&knn_graph = *host_knn_graph_ptr;
  knn_graph = new NNDElement[size * neighb_num];
  hipMemcpy(knn_graph, knn_graph_dev,
             (size_t)size * neighb_num * sizeof(NNDElement),
             hipMemcpyDeviceToHost);
}

void OutputHostKNNGraph(const vector<vector<NNDElement>> &knn_graph,
                        const string &out_path, const bool output_distance) {
  auto out = ofstream(out_path);
  if (!out.is_open()) {
    cerr << "Output file is not opened!" << endl;
    return;
  }
  out << knn_graph.size() << " " << knn_graph[0].size() << endl;
  for (int i = 0; i < knn_graph.size(); i++) {
    const auto &x = knn_graph[i];
    out << i << " " << x.size() << " ";
    for (auto y : x) {
      if (output_distance)
        out << y.distance() << " " << y.label() << "\t";
      else
        out << y.label() << "\t";
      // assert(y.label() != i);
    }
    out << endl;
  }
  out.close();
}

__global__ void GenRandKNNGraphIndexKernel(
    int *knn_graph_index, const int graph_size, const int neighb_num,
    const unsigned long long *random_sequence) {
  int list_id = blockIdx.x;
  int tx = threadIdx.x;
  int pos = list_id * neighb_num + tx;
  knn_graph_index[pos] = random_sequence[pos] % (unsigned long long)graph_size;
}

void GenerateRandomKNNGraphIndex(int **knn_graph_index_ptr,
                                 const int graph_size, const int neighb_num) {
  unsigned long long *random_sequence_dev;
  int *&knn_graph_index = *knn_graph_index_ptr;
  hipMalloc(&random_sequence_dev,
             graph_size * neighb_num * sizeof(unsigned long long));
  DevRNGLongLong(random_sequence_dev, graph_size * neighb_num);
  hipMalloc(&knn_graph_index, graph_size * neighb_num * sizeof(int));
  GenRandKNNGraphIndexKernel<<<graph_size, neighb_num>>>(
      knn_graph_index, graph_size, neighb_num, random_sequence_dev);
  hipDeviceSynchronize();
  hipFree(random_sequence_dev);
  auto cuda_status = hipGetLastError();
  if (cuda_status != hipSuccess) {
    cerr << "GenRandomKNNGraph failed: " << hipGetErrorString(cuda_status)
         << endl;
    exit(-1);
  }
}

size_t PredPeakGPUMemory(const int vecs_num, const int vecs_dim, const int k,
                         const int sample_num,
                         const bool thrust_random) {
  size_t nndescent_mem_cost =
      (size_t)vecs_num * vecs_dim * sizeof(float) +
      (size_t)vecs_num * (sample_num * 2) * sizeof(int) +
      (size_t)vecs_num * sizeof(int) + (size_t)vecs_num * sizeof(int) +
      (size_t)vecs_num * (sample_num * 2) * sizeof(int) +
      (size_t)vecs_num * sizeof(int) + (size_t)vecs_num * sizeof(int) +
      (size_t)vecs_num * k * sizeof(NNDElement) +
      max((size_t)vecs_num * k * sizeof(long long) *
              thrust_random,  // random_sequence
          (size_t)vecs_num * (k / 32 + (k % 32 != 0)) * sizeof(int)) +  // locks
      0;
  cout << "NNDescent GPU memory costs: " << nndescent_mem_cost / 1024 << " KB"
       << endl;
  return nndescent_mem_cost;
}

__host__ __device__ uint64_t xorshift64star(uint64_t x) {
  x += 42;
	x ^= x >> 12; // a
	x ^= x << 25; // b
	x ^= x >> 27; // c
	return x * 0x2545F4914F6CDD1D;
}